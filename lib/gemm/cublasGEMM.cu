#include "hip/hip_runtime.h"
#include "cublasGEMM.h"

#define MAX_STREAMS 1

__global__ void d2f_kernel(const double *din, float *dout, size_t dsize) {
	size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

__global__ void d2h_kernel(const double *din, half *dout, size_t dsize) {
	size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

// pitched memory address calculation.
// T* pElement = (T*)((char*)BaseAddress + Row * pitch) + Column;
// float* element = (float*)((char*)devPtr + r * pitch + c * sizeof(float));
__global__ void d2f_kernel_pitch_2D(const double *din, const size_t in_pitch, float *dout, const size_t out_pitch, const size_t nrows, const size_t ncols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if ((idx < ncols) && (idy < nrows))
	{
        double *in = (double *)((char*) din + idy * in_pitch); // in_pitch = 512
        float *out = (float *)((char*) dout + idy * out_pitch);
	    out[idx] = (float) in[idx];
	}
}

__global__ void d2h_kernel_pitch_2D(const double *din, const size_t in_pitch, half *dout, const size_t out_pitch, const size_t nrows, const size_t ncols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if ((idx < ncols) && (idy < nrows))
	{
        double *in = (double *)((char*) din + idy * in_pitch);
        half *out = (half *)((char*) dout + idy * out_pitch);
	    out[idx] = (half) in[idx];
    }
}

__global__ void d2f_kernel_pitch(const double *din, const size_t in_pitch, float *dout, const size_t out_pitch, const size_t dsize, const size_t ncols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < dsize)
	{
        size_t in_i = idx / ncols;
        size_t in_j = idx % ncols;

        size_t out_i = idx / ncols;
        size_t out_j = idx % ncols;
        double *in = (double *)((char*) din + in_i * in_pitch); // in_pitch = 512
        float *out = (float *)((char*) dout + out_i * out_pitch);

        out[out_j] = in[in_j];
        // printf("%d: %lu %lu %lu %lu, %f %f\n", idx, in_i, in_j, out_i, out_j, in[out_j], out[in_j]);
	}
}

__global__ void h2f_kernel(half *din, float *dout, size_t dsize) {
	size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

__global__ void reduction_kernel_2D(float *in, size_t sub_m, size_t sub_n, size_t num_streams) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int i, res = 0;

	if ((idx < sub_n) && (idy < sub_m))
	{
        float *ptr = in + idy * sub_n + idx;
        for (i = 0; i < num_streams; i++) {
            res += ptr[i * sub_m * sub_n];
        }
        in[idy * sub_n + idx] = res;
	}
}

__global__ void reduction_kernel(float *in, size_t sub_m, size_t sub_n, size_t num_streams) {
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int i, res = 0;
    if (idx < sub_m * sub_n) {
        for (i = 0; i < num_streams; i++) {
            res += in[idx + i * sub_m * sub_n];
        }
        in[idx] = res;
    }
}

void tensor_blockGemmEx_async_v3(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c, hipDataType Atype, hipDataType Btype, hipDataType Ctype, hipDataType computetype) {
    size_t i, j, k;
    size_t cross_row = x * sub_k, cross_col = sub_m * sub_k;
    float alpha = 1.0;
    float beta = 1.0;

    struct timeval h_start, h_end;
    unsigned long long h2d_time = 0, d2h_time = 0, kernel_time = 0, reduction_time = 0;

    // think about a way to use the power of 2 as the number of streams
    size_t memory_usage = sizeof(double) * sub_m * sub_k + sizeof(double) * sub_k * sub_n + sizeof(float) * sub_m * sub_n;
    size_t num_streams = 32;
    num_streams = (num_streams < ((size_t) 8192 * 1048576 / (memory_usage))) ? num_streams : ((size_t) 8192 * 1048576 / (memory_usage));
    printf("num_streams: %lu\n", num_streams);
    size_t num_outstreams = 4;
    size_t num_instreams = 8;
    size_t thread_num = 1024;
    hipblasHandle_t handle[num_streams];
    hipStream_t stream[num_streams];
    for (i = 0; i < num_streams; i++) {
        hipblasCreate(handle + i);
        hipStreamCreate(stream + i);
        hipblasSetMathMode(handle[i], HIPBLAS_TENSOR_OP_MATH);
        hipblasSetStream(handle[i], stream[i]);
    }

    int out_stream_index, in_stream_index;
    double *a_sub_d, *b_sub_d;
    float *c_sub_f;
    // here cannot exceed the GPU memory
    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k * num_streams);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n * num_streams);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n * num_streams);

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            out_stream_index = (j % num_outstreams) * num_instreams;
            hipMemsetAsync(c_sub_f + out_stream_index * sub_m * sub_n, 0, sizeof(float) * sub_m * sub_n * num_instreams, stream[out_stream_index]);
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                in_stream_index = (k % num_instreams) + out_stream_index;
                gettimeofday(&h_start, NULL);
                hipMemcpyAsync(a_sub_d + in_stream_index * sub_m * sub_k, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice, stream[in_stream_index]);    
                hipMemcpyAsync(b_sub_d + in_stream_index * sub_k * sub_n, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice, stream[in_stream_index]);
                gettimeofday(&h_end, NULL);
                h2d_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
                // async execution (ref: https://forums.developer.nvidia.com/t/async-cublas/2837)
                // hipDataType helps users to convert data inside the function call
                
                gettimeofday(&h_start, NULL);
                hipblasGemmEx(handle[in_stream_index], HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d + in_stream_index * sub_k * sub_n, Btype, sub_k, a_sub_d + in_stream_index * sub_m * sub_k, Atype, sub_m, &beta, c_sub_f + in_stream_index * sub_m * sub_n, Ctype, sub_m, computetype, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
                gettimeofday(&h_end, NULL);
                kernel_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
            }

            for (k = out_stream_index; k < num_instreams; k++) {
                hipStreamSynchronize(stream[k]);
            }

            if (num_streams > 1) {
                gettimeofday(&h_start, NULL);
                reduction_kernel<<<(sub_m*sub_n+thread_num-1)/thread_num, thread_num, 0, stream[out_stream_index]>>>(c_sub_f + out_stream_index * sub_m * sub_n, sub_m, sub_n, num_streams);
                gettimeofday(&h_end, NULL);
                reduction_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);                
            }
            gettimeofday(&h_start, NULL);
            hipMemcpyAsync((c + i * cross_row + j * cross_col), (c_sub_f + out_stream_index * sub_m * sub_n), sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost, stream[out_stream_index]);
            gettimeofday(&h_end, NULL);
            d2h_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
        }
    }

    hipDeviceSynchronize();
    for (i = 0; i < num_streams; i++) {
        hipblasDestroy(handle[i]);
        hipStreamDestroy(stream[i]);
    }
    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_f);
    printf("h2d time: %f ms\n", (float) h2d_time / 1000);
    printf("kernel time: %f ms\n", (float) kernel_time / 1000);
    printf("reduction time: %f ms\n", (float) reduction_time / 1000);
    printf("d2h time: %f ms\n", (float) d2h_time / 1000);
}

void tensor_blockGemmEx_async_v2(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c, hipDataType Atype, hipDataType Btype, hipDataType Ctype, hipDataType computetype) {
    size_t i, j, k;
    size_t cross_row = x * sub_k, cross_col = sub_m * sub_k;
    float alpha = 1.0;
    float beta = 1.0;
    double *a_sub_d, *b_sub_d;
    float *c_sub_f;
    struct timeval h_start, h_end;
    unsigned long long h2d_time = 0, d2h_time = 0, kernel_time = 0, reduction_time = 0;

    // think about a way to use the power of 2 as the number of streams
    size_t memory_usage = sizeof(double) * sub_m * sub_k + sizeof(double) * sub_k * sub_n + sizeof(float) * sub_m * sub_n;
    size_t num_streams = (MAX_STREAMS < ((size_t) 8192 * 1048576 / (memory_usage))) ? MAX_STREAMS : ((size_t) 8192 * 1048576 / (memory_usage));
    printf("num_streams: %lu\n", num_streams);

    hipblasHandle_t handle[num_streams];
    hipStream_t stream[num_streams];
    for (i = 0; i < num_streams; i++) {
        hipblasCreate(handle + i);
        hipStreamCreate(stream + i);
        hipblasSetMathMode(handle[i], HIPBLAS_TENSOR_OP_MATH);
        hipblasSetStream(handle[i], stream[i]);
    }

    int stream_index;
    
    // here cannot exceed the GPU memory
    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k * num_streams);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n * num_streams);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n * num_streams);

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            stream_index = j % num_streams;
            hipMemsetAsync(c_sub_f + stream_index * sub_m * sub_n, 0, sizeof(float) * sub_m * sub_n, stream[stream_index]);
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                gettimeofday(&h_start, NULL);
                hipMemcpyAsync(a_sub_d + stream_index * sub_m * sub_k, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice, stream[stream_index]);    
                hipMemcpyAsync(b_sub_d + stream_index * sub_k * sub_n, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice, stream[stream_index]);
                gettimeofday(&h_end, NULL);
                h2d_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
                // async execution (ref: https://forums.developer.nvidia.com/t/async-cublas/2837)
                // hipDataType helps users to convert data inside the function call
                
                gettimeofday(&h_start, NULL);
                hipblasGemmEx(handle[stream_index], HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d + stream_index * sub_k * sub_n, Btype, sub_k, a_sub_d + stream_index * sub_m * sub_k, Atype, sub_m, &beta, c_sub_f + stream_index * sub_m * sub_n, Ctype, sub_m, computetype, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
                gettimeofday(&h_end, NULL);
                kernel_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
            }
            // hipDeviceSynchronize();
            // if (num_streams > 1) {
            //     gettimeofday(&h_start, NULL);
            //     reduction_kernel<<<(sub_m*sub_n+thread_num-1)/thread_num, thread_num>>>(c_sub_f, sub_m, sub_n, num_streams);
            //     gettimeofday(&h_end, NULL);
            //     reduction_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);                
            // }
            gettimeofday(&h_start, NULL);
            hipMemcpyAsync((c + i * cross_row + j * cross_col), (c_sub_f + stream_index * sub_m * sub_n), sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost, stream[stream_index]);
            gettimeofday(&h_end, NULL);
            d2h_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
        }
    }
    for (i = 0; i < x; i++) {
        for (j = 0; j < y; j++) {
            printf("%f ", c[i*y+j]);
        }
        printf("\n");
    }  
    printf("\n");

    hipDeviceSynchronize();
    for (i = 0; i < num_streams; i++) {
        hipblasDestroy(handle[i]);
        hipStreamDestroy(stream[i]);
    }
    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_f);
    printf("h2d time: %f ms\n", (float) h2d_time / 1000);
    printf("kernel time: %f ms\n", (float) kernel_time / 1000);
    printf("reduction time: %f ms\n", (float) reduction_time / 1000);
    printf("d2h time: %f ms\n", (float) d2h_time / 1000);
}

// inner loop runs asynchronously
// assume the inputs are half precision now
void tensor_blockGemmEx_async(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c, hipDataType Atype, hipDataType Btype, hipDataType Ctype, hipDataType computetype) {
    size_t i, j, k;
    size_t cross_row = x * sub_k, cross_col = sub_m * sub_k;
    float alpha = 1.0;
    float beta = 1.0;
    half *a_sub_d, *b_sub_d;
    double *temp_a, *temp_b;
    float *c_sub_f;
    struct timeval h_start, h_end;
    unsigned long long h2d_time = 0, d2h_time = 0, kernel_time = 0, reduction_time = 0;

    size_t dsize = sub_m * sub_n;
    // think about a way to use the power of 2 as the number of streams
    size_t memory_usage = sizeof(double) * sub_m * sub_k + sizeof(double) * sub_k * sub_n + sizeof(float) * sub_m * sub_n;
    size_t num_streams = (MAX_STREAMS < ((size_t) 8192 * 1048576 / (memory_usage))) ? MAX_STREAMS : ((size_t) 8192 * 1048576 / (memory_usage));
    printf("num_streams: %lu\n", num_streams);

    hipMalloc((void **) &a_sub_d, sizeof(half) * num_streams * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(half) * num_streams * sub_k * sub_n);
    hipMalloc((void **) &c_sub_f, sizeof(float) * num_streams * sub_k * sub_n);    
    hipMalloc((void **) &temp_a, sizeof(double) * num_streams * sub_m * sub_k);
    hipMalloc((void **) &temp_b, sizeof(double) * num_streams * sub_k * sub_n);    

    hipblasHandle_t handle[num_streams];
    hipStream_t stream[num_streams];
    for (i = 0; i < num_streams; i++) {
        hipblasCreate(handle + i);
        hipStreamCreate(stream + i);
        hipblasSetMathMode(handle[i], HIPBLAS_TENSOR_OP_MATH);
        hipblasSetStream(handle[i], stream[i]);
    }

    // custom block gemm
    int stream_index;
    dim3 gridSize((sub_m+32-1)/32, (sub_n+32-1)/32);
    dim3 blockSize(32, 32);
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemset(c_sub_f, 0, sizeof(float) * sub_m * sub_n * num_streams);
            for (k = 0; k < (z / sub_k); k++) {
                stream_index = k % num_streams;
                // here we can use GPUDirect?
                gettimeofday(&h_start, NULL);
                hipMemcpyAsync(temp_a + stream_index * sub_m * sub_k, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice, stream[stream_index]);   
                d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK, 0, stream[stream_index]>>>(temp_a + stream_index * sub_m * sub_k, a_sub_d + stream_index * sub_m * sub_k, dsize);
                hipMemcpyAsync(temp_b + stream_index * sub_k * sub_n, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice, stream[stream_index]);
                d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK, 0, stream[stream_index]>>>(temp_b + stream_index * sub_m * sub_k, b_sub_d + stream_index * sub_m * sub_k, dsize);
                gettimeofday(&h_end, NULL);
                h2d_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
                // async execution (ref: https://forums.developer.nvidia.com/t/async-cublas/2837)
                // hipDataType helps users to convert data inside the function call
                gettimeofday(&h_start, NULL);
                hipblasGemmEx(handle[stream_index], HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d + stream_index * sub_m * sub_n, Btype, sub_k, a_sub_d + stream_index * sub_m * sub_n, Atype, sub_m, &beta, c_sub_f + stream_index * sub_m * sub_n, Ctype, sub_m, computetype, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
                gettimeofday(&h_end, NULL);
                kernel_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
            }
            hipDeviceSynchronize();
            if (num_streams > 1) {
                gettimeofday(&h_start, NULL);
                reduction_kernel_2D<<<gridSize, blockSize>>>(c_sub_f, sub_m, sub_n, num_streams);
                gettimeofday(&h_end, NULL);
                reduction_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);                
            }

            gettimeofday(&h_start, NULL);
            hipMemcpy((c + i * cross_row + j * cross_col), c_sub_f, sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost);
            gettimeofday(&h_end, NULL);
            d2h_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
        }
    }

    // for (i = 0; i < x; i++) {
    //     for (j = 0; j < y; j++) {
    //         printf("%f ", c[i*y+j]);
    //     }
    //     printf("\n");
    // }  
    // printf("\n");

    for (i = 0; i < num_streams; i++) {
        hipblasDestroy(handle[i]);
        hipStreamDestroy(stream[i]);
    }

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_f);
    hipFree(temp_a);
    hipFree(temp_b);


    printf("h2d time: %f ms\n", (float) h2d_time / 1000);
    printf("kernel time: %f ms\n", (float) kernel_time / 1000);
    printf("reduction time: %f ms\n", (float) reduction_time / 1000);
    printf("d2h time: %f ms\n", (float) d2h_time / 1000);
}

void tensor_blockGemmEx(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c, hipDataType Atype, hipDataType Btype, hipDataType Ctype, hipDataType computetype) {
    size_t i, j, k;
    size_t cross_row = x * sub_k, cross_col = sub_m * sub_k;
    float alpha = 1.0;
    float beta = 1.0;
    void *a_sub_d, *b_sub_d;
    double *temp_a, *temp_b;

    float *c_sub_f;
    struct timeval h_start, h_end;
    // assume input/output arrays are the same size and square matrix now
    const size_t dsize = sub_m * sub_n;

    unsigned long long h2d_time = 0, d2h_time = 0, kernel_time = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    // hipStream_t stream[z / sub_k];
    // for (i = 0; i < z / sub_k; i++) {
    //     hipStreamCreate(stream + i);
    // }
    

    if (Atype == HIP_R_16F || Atype == HIP_R_32F) {
        hipMalloc((void **) &temp_a, sizeof(double) * sub_m * sub_k);
        hipMalloc((void **) &temp_b, sizeof(double) * sub_k * sub_n);    
        if (Atype == HIP_R_16F) {
            // printf("half type\n");
            hipMalloc((void **) &a_sub_d, sizeof(half) * sub_m * sub_k);
            hipMalloc((void **) &b_sub_d, sizeof(half) * sub_k * sub_n);
        } else {
            // printf("float type\n");
            hipMalloc((void **) &a_sub_d, sizeof(float) * sub_m * sub_k);
            hipMalloc((void **) &b_sub_d, sizeof(float) * sub_k * sub_n);
        }
    } else if (Atype == HIP_R_64F) {
        hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
        hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);    
    } else {
        printf("input type: %d is not supported\n", Atype);
        return;
    }

    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n);

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemset(c_sub_f, 0, sub_m * sub_n * sizeof(float));
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                gettimeofday(&h_start, NULL);
                if (Atype == HIP_R_16F || Atype == HIP_R_32F) {
                    hipMemcpy(temp_a, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                    hipMemcpy(temp_b, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                    if (Atype == HIP_R_16F) {
                        d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(temp_a, (half *) a_sub_d, dsize);
                        d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(temp_b, (half *) b_sub_d, dsize);
                    } else { 
                        d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(temp_a, (float *) a_sub_d, dsize);
                        d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(temp_b, (float *) b_sub_d, dsize);
                    }
                } else { // HIP_R_64F
                    hipMemcpy(a_sub_d, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                    hipMemcpy(b_sub_d, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                } 
                gettimeofday(&h_end, NULL);
                h2d_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
                // async execution (ref: https://forums.developer.nvidia.com/t/async-cublas/2837)
                // hipDataType helps users to convert data inside the function call
                
                gettimeofday(&h_start, NULL);
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, Btype, sub_k, a_sub_d, Atype, sub_m, &beta, c_sub_f, Ctype, sub_m, computetype, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
                hipDeviceSynchronize();
                gettimeofday(&h_end, NULL);
                kernel_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
            }
            gettimeofday(&h_start, NULL);
            hipMemcpy((c + i * cross_row + j * cross_col), c_sub_f, sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost);
            gettimeofday(&h_end, NULL);
            d2h_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
        }
    }

    hipblasDestroy(handle);
    // for (i = 0; i < z / sub_k; i++) {
    //     hipStreamDestroy(stream[i]);
    // }

    if (Atype == HIP_R_16F || Atype == HIP_R_32F) {
        hipFree(temp_a);
        hipFree(temp_b);
    } 
    
    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_f);
    printf("h2d time: %f ms\n", (float) h2d_time / 1000);
    printf("kernel time: %f ms\n", (float) kernel_time / 1000);
    printf("d2h time: %f ms\n", (float) d2h_time / 1000);
}

void tensor_blockSgemm_half_async_v3(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    tensor_blockGemmEx_async_v3(x, y, z, sub_m, sub_n, sub_k, a, b, c, HIP_R_16F, HIP_R_16F, HIP_R_32F, HIP_R_32F);
}

void tensor_blockSgemm_half_async_v2(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    tensor_blockGemmEx_async_v2(x, y, z, sub_m, sub_n, sub_k, a, b, c, HIP_R_16F, HIP_R_16F, HIP_R_32F, HIP_R_32F);
}

void tensor_blockSgemm_half_async(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    tensor_blockGemmEx_async(x, y, z, sub_m, sub_n, sub_k, a, b, c, HIP_R_16F, HIP_R_16F, HIP_R_32F, HIP_R_32F);
}

void tensor_blockSgemm_half(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    tensor_blockGemmEx(x, y, z, sub_m, sub_n, sub_k, a, b, c, HIP_R_16F, HIP_R_16F, HIP_R_32F, HIP_R_32F);
}

void tensor_blockSgemm(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    tensor_blockGemmEx(x, y, z, sub_m, sub_n, sub_k, a, b, c, HIP_R_32F, HIP_R_32F, HIP_R_32F, HIP_R_32F);
}

// DON'T USE. Lose precision somewhere.
float* tensor_blockHgemm(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    size_t i, j, k;
    size_t cross_row = x * sub_k, cross_col = sub_m * sub_k;
    half alpha = 1.0;
    half beta = 1.0;
    half *a_sub_h, *b_sub_h, *c_sub_h;
    double *a_sub_d, *b_sub_d;
    float *c_sub_d;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_d, sizeof(float) * sub_m * sub_n);
    hipMalloc((void **) &a_sub_h, sizeof(half) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_h, sizeof(half) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_h, sizeof(half) * sub_m * sub_n);

    size_t dsize = sub_m * sub_n;

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemset(c_sub_h, 0, sub_m * sub_n * sizeof(half));
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                hipMemcpy(a_sub_d, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                hipMemcpy(b_sub_d, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_sub_d, a_sub_h, dsize);
                d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_sub_d, b_sub_h, dsize);
                hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_h, sub_k, a_sub_h, sub_m, &beta, c_sub_h, sub_m);
            }
            h2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_sub_h, c_sub_d, dsize);
            hipMemcpy((c + i * cross_row + j * cross_col), c_sub_d, sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost);
        }
    }
    
    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    hipFree(a_sub_h);
    hipFree(b_sub_h);
    hipFree(c_sub_h);

    return c;
}

void tensor_blockDgemm(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {    
    size_t i, j, k;
    size_t cross_row = x * sub_k, cross_col = sub_m * sub_k;
    double alpha = 1.0;
    double beta = 1.0;
    double *a_sub_d, *b_sub_d, *c_sub_d;
    float *c_sub_f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_d, sizeof(double) * sub_m * sub_n);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n);

    size_t dsize = sub_m * sub_n;

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemset(c_sub_d, 0, sub_m * sub_n * sizeof(double));
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                hipMemcpy(a_sub_d, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                hipMemcpy(b_sub_d, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, sub_k, a_sub_d, sub_m, &beta, c_sub_d, sub_m);
            }
            d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_sub_d, c_sub_f, dsize);
            hipMemcpy((c + i * cross_row + j * cross_col), c_sub_f, sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    hipFree(c_sub_f);
}

void sequential_blockGemmEx(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c, hipDataType Atype, hipDataType Btype, hipDataType Ctype, hipDataType computetype) {
    size_t i, j, k, ii, kk, i_idx, k_idx;
    float alpha = 1.0;
    float beta = 1.0;
    void *a_sub_d, *b_sub_d;
    double *temp_a, *temp_b;
    float *c_sub_f;
    struct timeval h_start, h_end;
    unsigned long long h2d_time = 0, d2h_time = 0, kernel_time = 0;
    size_t a_in_pitch, converted_a_in_pitch;
    size_t b_in_pitch, converted_b_in_pitch;
    size_t out_pitch;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    dim3 gridSize((sub_m+32-1)/32, (sub_n+32-1)/32);
    dim3 blockSize(32, 32);

    // for copying data and running kernel asynchronously: https://devblogs.nvidia.com/how-overlap-data-transfers-cuda-cc/
    // hipStream_t stream[sub_m];
    // for (i = 0; i < sub_m; i++) {
    //     hipStreamCreate(stream + i);
    // }

    size_t lda;
    size_t ldb;
    size_t ldc;
    if (Atype == HIP_R_16F || Atype == HIP_R_32F) {
        hipMallocPitch((void **) &temp_a, &a_in_pitch, sizeof(double) * sub_k, sub_m);
        hipMallocPitch((void **) &temp_b, &b_in_pitch, sizeof(double) * sub_n, sub_k);    
        if (Atype == HIP_R_16F) {
            // hipMalloc((void **) &a_sub_d, sizeof(half) * sub_m * sub_k);
            // hipMalloc((void **) &b_sub_d, sizeof(half) * sub_k * sub_n);
            hipMallocPitch((void **) &a_sub_d, &converted_a_in_pitch, sizeof(half) * sub_k, sub_m);
            hipMallocPitch((void **) &b_sub_d, &converted_b_in_pitch, sizeof(half) * sub_n, sub_k);
            lda = converted_a_in_pitch / sizeof(half);
            ldb = converted_b_in_pitch / sizeof(half);
        } else {
            // hipMalloc((void **) &a_sub_d, sizeof(float) * sub_m * sub_k);
            // hipMalloc((void **) &b_sub_d, sizeof(float) * sub_k * sub_n);
            hipMallocPitch((void **) &a_sub_d, &converted_a_in_pitch, sizeof(float) * sub_k, sub_m);
            hipMallocPitch((void **) &b_sub_d, &converted_b_in_pitch, sizeof(float) * sub_n, sub_k);
            lda = converted_a_in_pitch / sizeof(float);
            ldb = converted_b_in_pitch / sizeof(float);
        }
    } else if (Atype == HIP_R_64F) {
        hipMallocPitch((void **) &a_sub_d, &a_in_pitch, sizeof(double) * sub_k, sub_m);
        hipMallocPitch((void **) &b_sub_d, &b_in_pitch, sizeof(double) * sub_n, sub_k);    
        lda = a_in_pitch / sizeof(double);
        ldb = b_in_pitch / sizeof(double);
    } else {
        printf("input type: %d is not supported\n", Atype);
        return;
    }

    hipMallocPitch((void **) &c_sub_f, &out_pitch, sizeof(float) * sub_n, sub_m);
    ldc = out_pitch / sizeof(float);

    printf("a pitch: %lu, b pitch: %lu\n", a_in_pitch, b_in_pitch);   
    printf("converted a pitch: %lu, b pitch: %lu\n", converted_a_in_pitch, converted_b_in_pitch);
    printf("out pitch size: %lu\n", out_pitch);
    printf("lda: %lu, ldb: %lu, ldc: %lu\n", lda, ldb, ldc);
    // printf("temp_a address: %p\n", temp_a);
    // printf("temp_b address: %p\n", temp_b);

    for (i = 0; i < x; i += sub_m) {
        for (j = 0; j < y; j += sub_n) {
            hipMemset2D(c_sub_f, out_pitch, 0, sub_n * sizeof(float), sub_m);
            for (k = 0; k < z; k += sub_k) {
                gettimeofday(&h_start, NULL);
                if (Atype == HIP_R_16F || Atype == HIP_R_32F) {
                    hipMemcpy2D(temp_a, a_in_pitch, (a + i * y + k), z * sizeof(double), sizeof(double) * sub_k, sub_m, hipMemcpyHostToDevice);
                    hipMemcpy2D(temp_b, b_in_pitch, (b + k * y + j), y * sizeof(double), sizeof(double) * sub_n, sub_k, hipMemcpyHostToDevice);
                    if (Atype == HIP_R_16F) {
                        d2h_kernel_pitch_2D<<<gridSize, blockSize>>>(temp_a, a_in_pitch, (half *) a_sub_d, converted_a_in_pitch, sub_m, sub_k);
                        d2h_kernel_pitch_2D<<<gridSize, blockSize>>>(temp_b, b_in_pitch, (half *) b_sub_d, converted_b_in_pitch, sub_k, sub_n);
                    } else { 
                        d2f_kernel_pitch_2D<<<gridSize, blockSize>>>(temp_a, a_in_pitch, (float *) a_sub_d, converted_a_in_pitch, sub_m, sub_k);
                        d2f_kernel_pitch_2D<<<gridSize, blockSize>>>(temp_b, b_in_pitch, (float *) b_sub_d, converted_b_in_pitch, sub_k, sub_n);
                    }
                } 
                else { // HIP_R_64F
                    hipMemcpy2D(a_sub_d, a_in_pitch, (a + i * y + k), z * sizeof(double), sizeof(double) * sub_k, sub_m, hipMemcpyHostToDevice);
                    hipMemcpy2D(b_sub_d, b_in_pitch, (b + k * y + j), y * sizeof(double), sizeof(double) * sub_n, sub_k, hipMemcpyHostToDevice);    
                } 
                gettimeofday(&h_end, NULL);
                h2d_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
                // hipblasDgemm EXPLANATION ------------------------------------------------
                // the memory layout is different from we know
                // a = [0 1; b = [3 2; 
                //      2 3]      1 0]
                // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
                // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
                // maybe that's because inside GPU it uses column major storage.
                gettimeofday(&h_start, NULL);
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, Btype, ldb, a_sub_d, Atype, lda, &beta, c_sub_f, Ctype, ldc, computetype, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
                hipDeviceSynchronize();
                gettimeofday(&h_end, NULL);
                kernel_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);            
            }
            gettimeofday(&h_start, NULL);
            hipMemcpy2D((c + i * y + j), y * sizeof(float), c_sub_f, out_pitch, sizeof(float) * sub_n, sub_m, hipMemcpyDeviceToHost);

            // for (ii = i, i_idx = 0; ii < (i + sub_n); ii++, i_idx++) {
            //     hipMemcpyAsync((c + ii * y + j), (c_sub_f + i_idx * sub_n), sub_n * sizeof(float), hipMemcpyDeviceToHost, stream[i_idx]);
            // }   
            gettimeofday(&h_end, NULL);
            d2h_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);              
        }
    }  
    
    for (i = 0; i < x; i++) {
        for (j = 0; j < y; j++) {
            printf("%f ", c[i*y+j]);
        }
        printf("\n");
    }  
    printf("\n");
    hipblasDestroy(handle);
    
    if (Atype == HIP_R_16F || Atype == HIP_R_32F) {
        hipFree(temp_a);
        hipFree(temp_b);
    } 

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_f);

    printf("h2d time: %f ms\n", (float) h2d_time / 1000);
    printf("kernel time: %f ms\n", (float) kernel_time / 1000);
    printf("d2h time: %f ms\n", (float) d2h_time / 1000);
}

void sequential_blockSgemm_half(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    sequential_blockGemmEx(x, y, z, sub_m, sub_n, sub_k, a, b, c, HIP_R_16F, HIP_R_16F, HIP_R_32F, HIP_R_32F);
}

void sequential_blockSgemm(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    sequential_blockGemmEx(x, y, z, sub_m, sub_n, sub_k, a, b, c, HIP_R_32F, HIP_R_32F, HIP_R_32F, HIP_R_32F);
}

void sequential_blockDgemm_2D(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    size_t i, j, k, ii, kk, i_idx, k_idx;
    double alpha = 1.0;
    double beta = 1.0;
    double *a_sub_d, *b_sub_d, *c_sub_d;
    float *c_sub_f;
    size_t a_in_pitch, b_in_pitch, out_f_pitch, out_d_pitch;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // hipMalloc((void **) &a_sub_d, sizeof(double) * sub_k * sub_m);
    // hipMalloc((void **) &b_sub_d, sizeof(double) * sub_n * sub_k);
    // hipMalloc((void **) &c_sub_d, sizeof(double) * sub_n * sub_m);

    // cause reshape
    hipMallocPitch((void **) &a_sub_d, &a_in_pitch, sizeof(double) * sub_k, sub_m);
    hipMallocPitch((void **) &b_sub_d, &b_in_pitch, sizeof(double) * sub_n, sub_k);
    hipMallocPitch((void **) &c_sub_d, &out_d_pitch, sizeof(double) * sub_n, sub_m);
    hipMallocPitch((void **) &c_sub_f, &out_f_pitch, sizeof(float) * sub_n, sub_m);

    printf("a_in_pitch size: %lu\n", a_in_pitch); 
    printf("b_in_pitch size: %lu\n", b_in_pitch);   
  
    printf("out_d_pitch size: %lu\n", out_d_pitch);
    printf("out_f_pitch size: %lu\n", out_f_pitch);

    size_t dsize = sub_m * sub_n;
    size_t lda = a_in_pitch / sizeof(double);
    size_t ldb = b_in_pitch / sizeof(double);
    size_t ldc_d = out_d_pitch / sizeof(double);
    size_t ldc_f = out_f_pitch / sizeof(float);
    printf("ldc_d: %lu, ldc_f: %lu\n", ldc_d, ldc_f);

    dim3 gridSize((sub_m+32-1)/32, (sub_n+32-1)/32);
    dim3 blockSize(32, 32);
    
    for (i = 0; i < x; i += sub_m) {
        for (j = 0; j < y; j += sub_n) {
            // printf("memset\n");
            hipMemset2D(c_sub_d, out_d_pitch, 0, sub_n * sizeof(double), sub_m);
            for (k = 0; k < z; k += sub_k) {
                // hipMemcpy2D(c_sub_d, out_d_pitch, (a + i*y + k), z * sizeof(double), sub_k * sizeof(double), sub_m, hipMemcpyHostToDevice);
                hipMemcpy2D(a_sub_d, a_in_pitch, (a + i*y + k), z * sizeof(double), sub_k * sizeof(double), sub_m, hipMemcpyHostToDevice);
                
                // hipMemcpy2D(c_sub_d, out_d_pitch, (b + k*y + j), y * sizeof(double), sub_n * sizeof(double), sub_k, hipMemcpyHostToDevice);
                hipMemcpy2D(b_sub_d, b_in_pitch, (b + k*y + j), y * sizeof(double), sub_n * sizeof(double), sub_k, hipMemcpyHostToDevice);

                // hipblasDgemm EXPLANATION ------------------------------------------------
                // the memory layout is different from we know
                // a = [0 1; b = [3 2; 
                //      2 3]      1 0]
                // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
                // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
                // maybe that's because inside GPU it uses column major storage.
                // hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, sub_k, a_sub_d, sub_m, &beta, c_sub_d, sub_m);
                hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, ldb, a_sub_d, lda, &beta, c_sub_d, ldc_d);
                // hipMemcpy2D((c_d + i*y + k), y * sizeof(double), c_sub_d, out_d_pitch, sub_n * sizeof(double), sub_m, hipMemcpyDeviceToHost);
                // hipMemcpy2D((c_d + k*y + j), y * sizeof(double), c_sub_d, out_d_pitch, sub_n * sizeof(double), sub_m, hipMemcpyDeviceToHost);
            }
            d2f_kernel_pitch_2D<<<gridSize, blockSize>>>(c_sub_d, out_d_pitch, c_sub_f, out_f_pitch, sub_m, sub_n);
            // d2f_kernel_pitch<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_sub_d, out_d_pitch, c_sub_f, out_f_pitch, dsize, sub_n);
            hipMemcpy2D((c + i*y + j), y * sizeof(float), c_sub_f, out_f_pitch, sub_n * sizeof(float), sub_m, hipMemcpyDeviceToHost);
        }
    }  

    // for (i = 0; i < x; i++) {
    //     for (j = 0; j < y; j++) {
    //         printf("%f ", c[i*y+j]);
    //     }
    //     printf("\n");
    // }  
    // printf("\n");
    // printf("conversion\n");
    // hipMemcpy(c, c_f, dsize * sizeof(float), hipMemcpyDeviceToHost);
    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    hipFree(c_sub_f);
}

void sequential_blockDgemm(size_t x, size_t y, size_t z, size_t sub_m, size_t sub_n, size_t sub_k, 
    const double *a, const double *b, float *c) {
    size_t i, j, k, ii, kk, i_idx, k_idx;
    double alpha = 1.0;
    double beta = 1.0;
    double *a_sub_d, *b_sub_d, *c_sub_d;
    float *c_sub_f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_d, sizeof(double) * sub_m * sub_n);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n);

    size_t dsize = sub_m * sub_n;

    for (i = 0; i < x; i += sub_m) {
        for (j = 0; j < y; j += sub_n) {
            hipMemset(c_sub_d, 0, sub_m * sub_n * sizeof(double));
            for (k = 0; k < z; k += sub_k) {
                for (ii = i, i_idx = 0; ii < (i + sub_m); ii++, i_idx++) {
                    hipMemcpy((a_sub_d + i_idx * sub_n), (a + ii*y + k), sub_k * sizeof(double), hipMemcpyHostToDevice);
                }

                for (kk = k, k_idx = 0; kk < (k + sub_k); kk++, k_idx++) {
                    hipMemcpy((b_sub_d + k_idx * sub_n), (b + kk * y + j), sub_n * sizeof(double), hipMemcpyHostToDevice);
                }
                // hipblasDgemm EXPLANATION ------------------------------------------------
                // the memory layout is different from we know
                // a = [0 1; b = [3 2; 
                //      2 3]      1 0]
                // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
                // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
                // maybe that's because inside GPU it uses column major storage.
                hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, sub_k, a_sub_d, sub_m, &beta, c_sub_d, sub_m);
            }
            d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_sub_d, c_sub_f, dsize);
            for (ii = i, i_idx = 0; ii < (i + sub_n); ii++, i_idx++) {
                hipMemcpy((c + ii * y + j), (c_sub_f + i_idx * sub_n), sub_n * sizeof(float), hipMemcpyDeviceToHost);
            }                
        }
    }  
    
    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    hipFree(c_sub_f);
}

// DON'T USE. Lose precision somewhere.
float* wholeMatrixHgemm(size_t m, size_t n, size_t k, const double *a, const double *b, float *c) {
    half alpha = 1.0;
    half beta = 0.0;
    double *a_d, *b_d;
    half *a_h, *b_h, *c_h;
    float *c_f;
    size_t dsize = m * n;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_d, sizeof(double) * m * k);
    hipMalloc((void **) &b_d, sizeof(double) * k * n);
    hipMalloc((void **) &a_h, sizeof(half) * m * k);
    hipMalloc((void **) &b_h, sizeof(half) * k * n);
    hipMalloc((void **) &c_h, sizeof(half) * k * n);
    hipMalloc((void **) &c_f, sizeof(float) * m * n);

    hipMemcpy(a_d, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(double) * k * n, hipMemcpyHostToDevice);

    d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_d, a_h, dsize);
    d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_d, b_h, dsize);

    hipFree(a_d);
    hipFree(b_d);
    // hipblasDgemm EXPLANATION ------------------------------------------------
    // the memory layout is different from we know
    // a = [0 1; b = [3 2; 
    //      2 3]      1 0]
    // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
    // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
    // maybe that's because inside GPU it uses column major storage.
    hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_h, k, a_h, m, &beta, c_h, m);
    hipFree(a_h);
    hipFree(b_h);
    h2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_h, c_f, dsize);
    hipMemcpy(c, c_f, sizeof(float) * m * n, hipMemcpyDeviceToHost);
    
    hipblasDestroy(handle);
    hipFree(c_h);
    hipFree(c_f);
    return c;
}

void wholeMatrix_GemmEx(size_t m, size_t n, size_t k, const double *a, const double *b, float *c, hipDataType Atype, hipDataType Btype, hipDataType Ctype, hipDataType computetype) {
    float alpha = 1.0;
    float beta = 0.0;
    void *a_d, *b_d;
    float *c_f;
    // assume input/output arrays are the same size and square matrix now
    const size_t dsize = m * n;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    // Find out we have to make type conversion first.
    // Assume Atype == Btype
    // arrays need to be converted only when input type is half or float 
    if (Atype == HIP_R_16F || Atype == HIP_R_32F) {
        double *temp_a, *temp_b;
        hipMalloc((void **) &temp_a, sizeof(double) * m * k);
        hipMalloc((void **) &temp_b, sizeof(double) * k * n);
        hipMemcpy(temp_a, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
        hipMemcpy(temp_b, b, sizeof(double) * k * n, hipMemcpyHostToDevice);  
        if (Atype == HIP_R_16F) {
            hipMalloc((void **) &a_d, sizeof(half) * m * k);
            hipMalloc((void **) &b_d, sizeof(half) * k * n);
            d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(temp_a, (half *) a_d, dsize);
            d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(temp_b, (half *) b_d, dsize);
        } else {
            hipMalloc((void **) &a_d, sizeof(float) * m * k);
            hipMalloc((void **) &b_d, sizeof(float) * k * n);    
            d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(temp_a, (float *) a_d, dsize);
            d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(temp_b, (float *) b_d, dsize);
        }
        hipFree(temp_a);
        hipFree(temp_b);
    } else if (Atype == HIP_R_64F) {
        hipMemcpy(a_d, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
        hipMemcpy(b_d, b, sizeof(double) * k * n, hipMemcpyHostToDevice);    
    } else {
        printf("input type: %d is not supported\n", Atype);
        return;
    }
    
    hipMalloc((void **) &c_f, sizeof(float) * m * n);

    // hipblasDgemm EXPLANATION ------------------------------------------------
    // the memory layout is different from we know
    // a = [0 1; b = [3 2; 
    //      2 3]      1 0]
    // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
    // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
    // maybe that's because inside GPU it uses column major storage.
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_f, HIP_R_16F, k, a_f, HIP_R_16F, m, &beta, c_f, HIP_R_32F, m);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_d, Btype, k, a_d, Atype, m, &beta, c_f, Ctype, m, computetype, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    
    hipFree(a_d);
    hipFree(b_d);

    hipMemcpy(c, c_f, sizeof(float) * m * n, hipMemcpyDeviceToHost);
    
    hipblasDestroy(handle);

    hipFree(c_f);
}

void wholeMatrix_Sgemm_half(size_t m, size_t n, size_t k, const double *a, const double *b, float *c) {
    wholeMatrix_GemmEx(m, n, k, a, b, c, HIP_R_16F, HIP_R_16F, HIP_R_32F, HIP_R_32F);
}

void wholeMatrix_Sgemm(size_t m, size_t n, size_t k, const double *a, const double *b, float *c) {
    wholeMatrix_GemmEx(m, n, k, a, b, c, HIP_R_32F, HIP_R_32F, HIP_R_32F, HIP_R_32F);
}

void wholeMatrix_Dgemm(size_t m, size_t n, size_t k, const double *a, const double *b, float *c) {
    double alpha = 1.0;
    double beta = 0.0;
    double *a_d, *b_d, *c_d;
    float *c_f;
    size_t dsize = m * n;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_d, sizeof(double) * m * k);
    hipMalloc((void **) &b_d, sizeof(double) * k * n);
    hipMalloc((void **) &c_d, sizeof(double) * m * n);

    hipMemcpy(a_d, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(double) * k * n, hipMemcpyHostToDevice);

    // hipblasDgemm EXPLANATION ------------------------------------------------
    // the memory layout is different from we know
    // a = [0 1; b = [3 2; 
    //      2 3]      1 0]
    // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
    // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
    // maybe that's because inside GPU it uses column major storage.
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_d, k, a_d, m, &beta, c_d, m);
    hipFree(a_d);
    hipFree(b_d);

    hipMalloc((void **) &c_f, sizeof(float) * m * n);
    d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_d, c_f, dsize);
    hipMemcpy(c, c_f, sizeof(float) * m * n, hipMemcpyDeviceToHost);
    
    hipblasDestroy(handle);

    hipFree(c_d);
    hipFree(c_f);
}