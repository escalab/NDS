#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>

// for mmap
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>

// for timing
#include <sys/time.h>

extern "C" {
    #include "spdkrpc.h"
}

#define RANDOMRESETPROB 0.15
#define THREADS_PER_BLOCK 1024

// void get_outedges(int64_t* graph, int64_t* outedges, size_t interval_st, size_t num_of_vertices_per_stripe, size_t num_of_vertices) {
//     memcpy(outedges, (graph + interval_st * num_of_vertices), STRIPE_SZ);
// }

// void get_inedges(int64_t* graph, int64_t* inedges, size_t interval_st, size_t interval_en, size_t num_of_vertices) {
//     size_t i, j, row;
//     int64_t *graph_ptr, *inedges_ptr;
//     // in column favor but transpose into row.
//     for (i = 0; i < num_of_vertices; i++) {
//         for (row = 0, j = interval_st; j < interval_en; row++, j++) {
//             // printf("i: %lu, j: %lu, row: %lu\n", i, j, row);
//             *(inedges + row * num_of_vertices + i) = *(graph + i * num_of_vertices + j);
//         }
//     }
// }

// construct vertices metadata first

__global__ void pagerank_update(double* prev_pr, double* curr_pr, double *vertices, size_t st, int64_t* inedges, int64_t *outedges, size_t num_of_vertices, size_t num_of_subvertices, size_t iter, size_t niters) {
    // v.outc is num_outedges()
    // needs: v.num_inedges(), v.inedge(), v.id(), v.outc, v.set_data
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (id >= num_of_subvertices) {
        return;
    }

    int64_t *inedge = inedges + id * num_of_vertices;
    int64_t *outedge = outedges + id;
    size_t i, outc = 0;
    double sum = 0;

    id = st + id;
    for (i = 0; i < num_of_vertices; i++) {
        if (i != id && outedge[i * num_of_subvertices] != 0) {
            outc++;
        }
    }

    // first iteration
    if (iter > 0) {
        for (i = 0; i < num_of_vertices; i++) {
            // we don't consider self-loop
            if (inedge[i] && i != id) {
                sum += prev_pr[i];
            }
        }
        if (outc > 0) {
            curr_pr[id] = (RANDOMRESETPROB + (1 - RANDOMRESETPROB) * sum) / (double) outc;
        } else {
            curr_pr[id] = (RANDOMRESETPROB + (1 - RANDOMRESETPROB) * sum);
        }
    } else if (iter == 0) {
        if (outc > 0) {
            curr_pr[id] = 1.0f / (double) outc;
        }
    }

    // for the last iteration
    if (iter == niters - 1) {
        if (outc > 0) {
            vertices[id] = curr_pr[id] * (double) outc;
        } else {
            vertices[id] = curr_pr[id];
        }
    }
}

/**
 * TODO section:
 * maybe we can borrow the log system from graphchi?
 */
int main(int argc, char** argv) {
    int id, rc;
    size_t num_of_vertices, num_of_subvertices, niters;
    size_t return_size, stripe_size, iter, st, i;
    
    int64_t *graph, *outedges, *inedges;
    struct JSONRPCClient client;

    // result
    double *vertices;
    double *prev_pr_d, *curr_pr_d, *vertices_d;

    // timing
    struct timeval h_start, h_end;
    uint64_t fetch_row_time = 0, fetch_col_time = 0, kernel_time = 0;

    if (argc < 5) {
        printf("usage: %s <matrix id> <# of vertices> <# of subvertices> <niters>\n", argv[0]);
        exit(1);
    } 

    rc = connect_to_spdkrpc_server(&client);
    if (rc) {
        printf("cannot create conntection to SPDK RPC server");
        return rc;
    }

    graph = (int64_t *) mmap_to_tensorstore_hugepage();
    if (graph == NULL) {
        return -1;
    }

    id = atoi(argv[1]);
    num_of_vertices = (size_t) atoi(argv[2]);
    num_of_subvertices = (size_t) atoi(argv[3]);
    niters = (size_t) atoi(argv[4]);
    
    stripe_size = num_of_vertices * num_of_subvertices * sizeof(int64_t);
    // subgraph initialization
    hipMalloc((void **) &outedges, stripe_size);
    hipMalloc((void **) &inedges, stripe_size);

    // PR initialization
    vertices = (double *) malloc(sizeof(double) * num_of_vertices);
    for (i = 0; i < num_of_vertices; i++) {
        vertices[i] = RANDOMRESETPROB;
    }

    hipMalloc((void **) &vertices_d, sizeof(double) * num_of_vertices);
    hipMalloc((void **) &prev_pr_d, sizeof(double) * num_of_vertices);
    hipMalloc((void **) &curr_pr_d, sizeof(double) * num_of_vertices);
    hipMemset(vertices_d, 0, sizeof(double) * num_of_vertices);
    hipMemcpy(prev_pr_d, vertices, sizeof(double) * num_of_vertices, hipMemcpyHostToDevice);
    hipMemset(curr_pr_d, 0, sizeof(double) * num_of_vertices);

    memset(vertices, 0, sizeof(double) * num_of_vertices);
    // Kernel loop (inspired by GraphChi engine)
    for (iter = 0; iter < niters; iter++) {
        printf("iter: %lu\n", iter);
        // userprogram.before_iteration(iter, chicontext);
        // do nothing in example

        // scheduler function
        // do nothing in example

        // shuffle function
        // do nothing in example

        // Interval loop
        // assume we have no subinterval in an interval.
        for (st = 0; st < (num_of_vertices / num_of_subvertices); st++) {
            /* preprocessing */

            // userprogram.before_exec_interval(interval_st, interval_en, chicontext);
            // do nothing in example

            // flush things back from sliding_shards

            // create a new memory shard
            gettimeofday(&h_start, NULL);
            return_size = tensorstore_get_row_stripe_submatrix(&client, id, st, st+1, num_of_subvertices);
            hipMemcpy(inedges, graph, return_size, hipMemcpyHostToDevice);
            gettimeofday(&h_end, NULL);
            fetch_row_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);   
            // get_outedges(graph, outedges, st, num_of_vertices_per_stripe, num_of_vertices);
            
            gettimeofday(&h_start, NULL);
            return_size = tensorstore_get_col_stripe_submatrix(&client, id, st, st+1, num_of_subvertices);
            hipMemcpy(outedges, graph, return_size, hipMemcpyHostToDevice);
            gettimeofday(&h_end, NULL);
            fetch_col_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);   
            // inedges need to be transposed (?)
            // get_inedges(graph, inedges, st, st + num_of_vertices_per_stripe, num_of_vertices);

            // initialize vertices & edge data

            // load data
            // load_before_updates(vertices);

            /* execution part */

            // exec_updates(userprogram, vertices);
            // exec_updates(GraphChiProgram<VertexDataType, EdgeDataType, svertex_t> &userprogram, std::vector<svertex_t> &vertices)
            
            // update vertices one by one
            // userprogram.update(v, chicontext);
            // update(graphchi_vertex<VertexDataType, EdgeDataType> &v, graphchi_context &ginfo);
            printf("st: %lu\n", st * num_of_subvertices);
            gettimeofday(&h_start, NULL);
            pagerank_update<<<(num_of_subvertices+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(prev_pr_d, curr_pr_d, vertices_d, st * num_of_subvertices, inedges, outedges, num_of_vertices, num_of_subvertices, iter, niters);
            gettimeofday(&h_end, NULL);
            kernel_time += ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);   

            /* postprocessing */

            // save_vertices(vertices);
            // nothing is modified in PR application

            // memoryshard->commit(modifies_inedges, modifies_outedges & !disable_outedges);
            // memoryshard->commit(0, 0) in PR. doesn't need to commit back

            // userprogram.after_exec_interval(interval_st, interval_en, chicontext);
            // do nothing in PR
        }
        hipMemcpy(prev_pr_d, curr_pr_d, sizeof(double) * num_of_vertices, hipMemcpyDeviceToDevice);
        // userprogram.after_iteration(iter, chicontext);
        // do nothing in PR

        // flush all stuff in sliding_shards
        // nothing inside maybe. it is just a buffer for updating
    }
    hipMemcpy(vertices, vertices_d, sizeof(double) * num_of_vertices, hipMemcpyDeviceToHost);

    FILE *fp = fopen("log.txt", "w");
    for (i = 0; i < num_of_vertices; i++) {
        fprintf(fp, "%lu %f\n", i, vertices[i]);
    }

    printf("row fetch time: %f ms\n", (float) fetch_row_time / 1000);
    printf("col fetch time: %f ms\n", (float) fetch_col_time / 1000);
    printf("kernel time: %f ms\n", (float) kernel_time / 1000);

    fclose(fp);
    // cleanup
    munmap(graph, HUGEPAGE_SZ);
    hipFree(outedges);
    hipFree(inedges);
    hipFree(vertices_d);
    hipFree(prev_pr_d);
    hipFree(curr_pr_d);
    free(vertices);

    return 0;
}