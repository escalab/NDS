#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// for mmap
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>

// for timing
#include <sys/time.h>

#include <hipblas.h>

#define THREADS_PER_BLOCK 256
#define WARMUP 1
#define ITERATIONS 10

__global__ void d2f_kernel(const double *din, float *dout, size_t dsize) {
	size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

__global__ void f2h_kernel(const float *din, half *dout, size_t dsize) {
	size_t idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

int isfinite_arr(double *ptr, size_t n) {
    size_t i;

    for (i = 0; i < n; i++) {
        if (!isfinite(ptr[i])) {
            return 0; 
        }
    }
    return 1;
}

int isfinite_arr(float *ptr, size_t n) {
    size_t i;
    for (i = 0; i < n; i++) {
        if (!isfinite(ptr[i])) {
            return 0; 
        }
    }
    return 1;
}

void test_loop(size_t n, hipblasOperation_t a_op, hipblasOperation_t b_op, 
    void *a, hipDataType a_type, void *b, hipDataType b_type,  
    void *c, hipDataType c_type, hipDataType compute_type) {
    int i;
    float alpha = 1.0;
    float beta = 0.0;

    struct timeval h_start, h_end;
    long duration;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    for (i = 0; i < WARMUP; i++) {
        hipblasGemmEx(handle, a_op, b_op, n, n, n, &alpha, a, a_type, n, b, b_type, n, &beta, c, c_type, n, compute_type, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        hipDeviceSynchronize();
    }

    for (i = 0; i < ITERATIONS; i++) {
        gettimeofday(&h_start, NULL);
        hipblasGemmEx(handle, a_op, b_op, n, n, n, &alpha, a, a_type, n, b, b_type, n, &beta, c, c_type, n, compute_type, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        hipDeviceSynchronize();
        gettimeofday(&h_end, NULL);
        duration = ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);
        printf("%.3f ", (float) duration / 1000);
    }
    printf("\n");
    hipblasDestroy(handle);
}

int main(int argc, char** argv) {
    double *a, *b;
    double *a_d, *b_d, *c_d, *c_host_d;
    float *a_f, *b_f, *c_f, *c_host_f;
    half *a_h, *b_h;
    float *c_h, *c_host_h;
    size_t n, dsize;
    int a_fd, b_fd;
    int i, j;
    hipblasOperation_t op_arr[] = {HIPBLAS_OP_N, HIPBLAS_OP_T};

    if (argc < 4) {
        printf("usage: %s <matrix A path> <matrix B path> <matrix size>\n", argv[0]);
        exit(1);
    }

    // GEMM configuration.
    a_fd = open(argv[1], O_RDONLY);
    b_fd = open(argv[2], O_RDONLY);
    n = atoi(argv[3]);
    dsize = n * n;

    a = (double *) mmap(NULL, sizeof(double) * n * n, PROT_READ, MAP_PRIVATE, a_fd, 0);
    b = (double *) mmap(NULL, sizeof(double) * n * n, PROT_READ, MAP_PRIVATE, b_fd, 0);
    hipMalloc((void **) &a_d, sizeof(double) * n * n);
    hipMalloc((void **) &b_d, sizeof(double) * n * n);
    hipMalloc((void **) &c_d, sizeof(double) * n * n);

    hipMemcpy(a_d, a, sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(double) * n * n, hipMemcpyHostToDevice);

    munmap(a, sizeof(double) * n * n);
    munmap(b, sizeof(double) * n * n);
    close(a_fd);
    close(b_fd);

    printf("memory usage: input: %lu bytes, output: %lu bytes\n", sizeof(double) * n * n * 2, sizeof(double) * n * n);
    
    // double
    c_host_d = (double *) malloc(sizeof(double) * n * n);
    printf("Running double precision...\n");
    for (i = 0; i < 2; i++) {
        for (j = 0; j < 2; j++) {
            test_loop(n, op_arr[i], op_arr[j], a_d, HIP_R_64F, b_d, HIP_R_64F, c_d, HIP_R_64F, HIP_R_64F);
            hipMemcpy(c_host_d, c_d, sizeof(double) * n * n, hipMemcpyDeviceToHost);
            if (!isfinite_arr(c_host_d, dsize)) {
                printf("found non-finite result, abort\n");
                exit(-1);
            }
        }
    }
    free(c_host_d);

    // float
    hipFree(c_d);
    hipMalloc((void **) &a_f, sizeof(float) * n * n);
    d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_d, a_f, dsize);
    hipFree(a_d);
    hipMalloc((void **) &b_f, sizeof(float) * n * n);
    d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_d, b_f, dsize);
    hipFree(b_d);
    hipMalloc((void **) &c_f, sizeof(float) * n * n);

    printf("Running float precision...\n");
    c_host_f = (float *) malloc(sizeof(float) * n * n);
    for (i = 0; i < 2; i++) {
        for (j = 0; j < 2; j++) {
            test_loop(n, op_arr[i], op_arr[j], a_f, HIP_R_32F, b_f, HIP_R_32F, c_f, HIP_R_32F, HIP_R_32F);
            hipMemcpy(c_host_f, c_f, sizeof(float) * n * n, hipMemcpyDeviceToHost);
            if (!isfinite_arr(c_host_f, dsize)) {
                printf("found non-finite result, abort\n");
                exit(-1);
            }
        }
    }
    free(c_host_f);

    // half
    hipFree(c_f);
    hipMalloc((void **) &a_h, sizeof(half) * n * n);
    f2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_f, a_h, dsize);
    hipFree(a_f);
    hipMalloc((void **) &b_h, sizeof(half) * n * n);
    f2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_f, b_h, dsize);
    hipFree(b_f);
    hipMalloc((void **) &c_h, sizeof(float) * n * n);

    printf("Running half precision...\n");
    c_host_h = (float *) malloc(sizeof(float) * n * n);
    for (i = 0; i < 2; i++) {
        for (j = 0; j < 2; j++) {
            test_loop(n, op_arr[i], op_arr[j], a_h, HIP_R_16F, b_h, HIP_R_16F, c_h, HIP_R_32F, HIP_R_32F);
            hipMemcpy(c_host_h, c_h, sizeof(float) * n * n, hipMemcpyDeviceToHost);
            if (!isfinite_arr(c_host_h, dsize)) {
                printf("found non-finite result, abort\n");
                exit(-1);
            }
        }
    }
    free(c_host_h);
    hipFree(a_h);
    hipFree(b_h);
    hipFree(c_h);

    return 0;
}