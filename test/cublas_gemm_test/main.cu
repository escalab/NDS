#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

// for mmap
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>

void verify(double *A, double *B, float *C, unsigned int m, unsigned int n, unsigned int k) {
    const float relativeTolerance = 1e-3;
  
    for(int row = 0; row < m; ++row) {
        for(int col = 0; col < n; ++col) {
            float sum = 0;
            for(unsigned int i = 0; i < k; ++i) {
                // printf("C[%u][%u] = A[%u][%u] * B[%u][%u]\n", row, col, row, i, i, col);
                sum += (float) A[row*k + i]*B[i*n + col];
            }
            float relativeError = (sum - C[row*n + col])/sum;
            if (fabs(relativeError) > relativeTolerance) {
                printf("(%d, %d) = %f, supposed to be %f\n", row, col, C[row*n + col], sum); 
                printf("TEST FAILED\n\n");
                exit(0);
            }
        }
    }
    printf("TEST PASSED\n\n");
}

float* doMultiply2Matrices(int m, int n, int k, double *a, double *b, float *c) {
    double alpha = 1.0;
    double beta = 0.0;
    double *a_d, *b_d, *c_d, *c_h;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_d, sizeof(double) * m * k);
    hipMalloc((void **) &b_d, sizeof(double) * k * n);
    hipMalloc((void **) &c_d, sizeof(double) * m * n);
    c_h = (double *) malloc(sizeof(double) * m * n);

    hipMemcpy(a_d, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(double) * k * n, hipMemcpyHostToDevice);

    // the memory layout is different from we know
    // a = [0 1; b = [3 2; 
    //      2 3]      1 0]
    // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
    // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
    // maybe that's because inside GPU it uses column major storage.
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_d, k, a_d, m, &beta, c_d, m);
    
    hipMemcpy(c_h, c_d, sizeof(double) * m * n, hipMemcpyDeviceToHost);
    hipblasDestroy(handle);

    for (int i = 0; i < m * n; ++i) {
        c[i] = (float) c_h[i];
    }

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(c_h);
    return c;
}


int main(int argc, char** argv) {
    double *a, *b;
    float *c;
    int n;
    int a_fd, b_fd;

    if (argc < 3) {
        printf("usage: %s <matrix path> <matrix size>\n", argv[0]);
        exit(1);
    }

    // GEMM configuration.
    a_fd = open(argv[1], O_RDONLY);
    b_fd = open(argv[1], O_RDONLY);

    n = atoi(argv[2]);
    a = (double *) mmap(NULL, sizeof(double) * n * n, PROT_READ, MAP_PRIVATE, a_fd, 0);
    b = (double *) mmap(NULL, sizeof(double) * n * n, PROT_READ, MAP_PRIVATE, b_fd, 0);
  
    // a = (double *) malloc(sizeof(double) * n * n);
    // b = (double *) malloc(sizeof(double) * n * n);

    c = (float *) calloc(n * n, sizeof(float));

    doMultiply2Matrices(n, n, n, a, b, c);

#ifdef DEBUG
    int i, j;
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", a[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", b[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", c[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
#endif
    verify(a, b, c, n, n, n);
    munmap(a, sizeof(double) * n * n);
    munmap(b, sizeof(double) * n * n);
    close(a_fd);
    close(b_fd);
    free(c);
    return 0;
}