#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

// for mmap
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>

float* tensor_blockmm(int x, int y, int z, int sub_m, int sub_n, int sub_k, 
    double *a, double *b, float *c) {
    int i, j, k;
    int cross_row = x * sub_k, cross_col = sub_m * sub_k;
    double alpha = 1.0;
    double beta = 1.0;
    double *a_sub_d, *b_sub_d, *c_sub_d;
    double *c_h;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    c_h = (double *) calloc(x * y, sizeof(double));

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_d, sizeof(double) * sub_m * sub_n);

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemcpy(c_sub_d, (c_h + i * cross_row + j * cross_col), sub_m * sub_n * sizeof(double), hipMemcpyHostToDevice);
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                hipMemcpy(a_sub_d, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                hipMemcpy(b_sub_d, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, sub_k, a_sub_d, sub_m, &beta, c_sub_d, sub_m);
            }
            hipMemcpy((c_h + i * cross_row + j * cross_col), c_sub_d, sub_m * sub_n * sizeof(double), hipMemcpyDeviceToHost);
        }
    }
    
    hipblasDestroy(handle);

    for (int i = 0; i < x * y; ++i) {
        c[i] = (float) c_h[i];
    }

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    free(c_h);

    return c;
}

float* sequential_blockmm(int x, int y, int z, int sub_m, int sub_n, int sub_k, 
    double *a, double *b, float *c) {
    int i, j, k, ii, jj, kk, i_idx, j_idx, k_idx;
    double alpha = 1.0;
    double beta = 0.0;
    double *a_sub_d, *b_sub_d, *c_sub_d;
    double *a_sub_h, *b_sub_h, *c_sub_h;
    double *c_h;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    a_sub_h = (double *) malloc(sizeof(double) * sub_m * sub_k);
    b_sub_h = (double *) malloc(sizeof(double) * sub_k * sub_n);
    c_sub_h = (double *) malloc(sizeof(double) * sub_m * sub_n);
    c_h = (double *) malloc(sizeof(double) * x * y);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_d, sizeof(double) * sub_m * sub_n);

    for(i = 0; i < x; i += sub_m) {
        for(j = 0; j < y; j += sub_n) {
            for(k = 0; k < z; k += sub_k) {
                // for (ii = i, i_idx = 0; ii < (i + sub_m); ii++, i_idx++) {
                //     for (jj = j, j_idx = 0; jj < (j + sub_n); jj++, j_idx++) {
                //         c_sub_h[i_idx * sub_n + j_idx] = c_h[ii * y + jj];
                //     }
                // }
                for (ii = i, i_idx = 0; ii < (i + sub_m); ii++, i_idx++) {
                    for (kk = k, k_idx = 0; kk < (k + sub_k); kk++, k_idx++) {
                        a_sub_h[i_idx * sub_n + k_idx] = a[ii*y + kk];         
                    }
                }

                for (jj = j, j_idx = 0; jj < (j + sub_n); jj++, j_idx++) {
                    for (kk = k, k_idx = 0; kk < (k + sub_k); kk++, k_idx++) {
                        b_sub_h[k_idx * sub_n + j_idx] = b[kk * y + jj];
                    }
                }
                hipMemcpy(a_sub_d, a_sub_h, sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);
                hipMemcpy(b_sub_d, b_sub_h, sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                // cudaMemcpy(c_sub_d, c_sub_h, x * y * sizeof(double), cudaMemcpyHostToDevice);
                // cublasDgemm EXPLANATION ------------------------------------------------
                // the memory layout is different from we know
                // a = [0 1; b = [3 2; 
                //      2 3]      1 0]
                // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
                // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
                // maybe that's because inside GPU it uses column major storage.
                hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, sub_k, a_sub_d, sub_m, &beta, c_sub_d, sub_m);
                hipMemcpy(c_sub_h, c_sub_d, sub_m * sub_n * sizeof(double), hipMemcpyDeviceToHost);

                for (ii = i, i_idx = 0; ii < (i + sub_n); ii++, i_idx++) {
                    for (jj = j, j_idx = 0; jj < (j + sub_n); jj++, j_idx++) {
                        // could be casted to double here?
                        c_h[ii * y + jj] += c_sub_h[i_idx * sub_n + j_idx];
                    }
                }  
            }              
        }
    }  
    
    hipblasDestroy(handle);

    for (int i = 0; i < x * y; ++i) {
        c[i] = (float) c_h[i];
    }

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    free(a_sub_h);
    free(b_sub_h);
    free(c_sub_h);
    free(c_h);

    return c;
}

float* doMultiply2Matrices(int m, int n, int k, const double *a, const double *b, float *c) {
    double alpha = 1.0;
    double beta = 0.0;
    double *a_d, *b_d, *c_d, *c_h;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_d, sizeof(double) * m * k);
    hipMalloc((void **) &b_d, sizeof(double) * k * n);
    hipMalloc((void **) &c_d, sizeof(double) * m * n);
    c_h = (double *) malloc(sizeof(double) * m * n);

    hipMemcpy(a_d, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(double) * k * n, hipMemcpyHostToDevice);

    // cublasDgemm EXPLANATION ------------------------------------------------
    // the memory layout is different from we know
    // a = [0 1; b = [3 2; 
    //      2 3]      1 0]
    // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
    // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
    // maybe that's because inside GPU it uses column major storage.
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_d, k, a_d, m, &beta, c_d, m);
    
    hipMemcpy(c_h, c_d, sizeof(double) * m * n, hipMemcpyDeviceToHost);
    hipblasDestroy(handle);

    for (int i = 0; i < m * n; ++i) {
        c[i] = (float) c_h[i];
    }

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(c_h);
    return c;
}

void cpu_verify(double *A, double *B, float *C, unsigned int m, unsigned int n, unsigned int k) {
    const float relativeTolerance = 1e-3;
  
    for(int row = 0; row < m; ++row) {
        for(int col = 0; col < n; ++col) {
            float sum = 0;
            for(unsigned int i = 0; i < k; ++i) {
                // printf("C[%u][%u] = A[%u][%u] * B[%u][%u]\n", row, col, row, i, i, col);
                sum += (float) A[row*k + i]*B[i*n + col];
            }
            float relativeError = (sum - C[row*n + col])/sum;
            if (fabs(relativeError) > relativeTolerance) {
                printf("(%d, %d) = %f, supposed to be %f\n", row, col, C[row*n + col], sum); 
                printf("TEST FAILED\n\n");
                exit(0);
            }
        }
    }
    printf("TEST PASSED\n\n");
}

void gpu_verify(const double *A, const double *B, float *C, unsigned int m, unsigned int n, unsigned int k) {
    const float relativeTolerance = 1e-3;
    float *c_valid = (float *) malloc(sizeof(float) * m * n);
    doMultiply2Matrices(m, n, k, A, B, c_valid);

    for(int row = 0; row < m; ++row) {
        for(int col = 0; col < n; ++col) {
            float relativeError = (c_valid[row*n + col] - C[row*n + col]) / c_valid[row*n + col];
            if (fabs(relativeError) > relativeTolerance) {
                printf("(%d, %d) = %f, supposed to be %f\n", row, col, C[row*n + col], c_valid[row*n + col]); 
                printf("TEST FAILED\n\n");
                exit(0);
            }
        }
    }
    printf("TEST PASSED\n\n");
    free(c_valid);
}


int main(int argc, char** argv) {
    double *a, *b;
    float *c;
    int n, sub_n;
    int a_fd, b_fd;

    if (argc < 3) {
        printf("usage: %s <sequence format path> <tensor format path> <matrix size> <submatrix size>\n", argv[0]);
        exit(1);
    }

    // GEMM configuration.
    a_fd = open(argv[1], O_RDONLY);
    b_fd = open(argv[1], O_RDONLY);

    n = atoi(argv[3]);
    sub_n = atoi(argv[4]);

    a = (double *) mmap(NULL, sizeof(double) * n * n, PROT_READ, MAP_PRIVATE, a_fd, 0);
    b = (double *) mmap(NULL, sizeof(double) * n * n, PROT_READ, MAP_PRIVATE, b_fd, 0);
  
    // a = (double *) malloc(sizeof(double) * n * n);
    // b = (double *) malloc(sizeof(double) * n * n);

    c = (float *) calloc(n * n, sizeof(float));
    // doMultiply2Matrices(n, n, n, a, b, c);
    sequential_blockmm(n, n, n, sub_n, sub_n, sub_n, a, b, c);


#ifdef DEBUG
    int i, j;
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", a[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", b[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", c[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
#endif
    gpu_verify(a, b, c, n, n, n);

    // GEMM configuration.
    int a_tensor_fd = open(argv[2], O_RDONLY);
    int b_tensor_fd = open(argv[2], O_RDONLY);

    double *a_tensor = (double *) mmap(NULL, sizeof(double) * n * n, PROT_READ, MAP_PRIVATE, a_tensor_fd, 0);
    double *b_tensor = (double *) mmap(NULL, sizeof(double) * n * n, PROT_READ, MAP_PRIVATE, b_tensor_fd, 0);
    
    memset(c, 0, sizeof(float) * n * n);

    tensor_blockmm(n, n, n, sub_n, sub_n, sub_n, a_tensor, b_tensor, c);

    printf("Reformat from tensor to sequential...\n");
    
    int count = 0;
    float *c_reformat = (float *) calloc(n * n, sizeof(float));
    for (int i = 0; i < n; i += sub_n) {
        for (int j = 0; j < n; j += sub_n) {  
            for(int ii = i; ii < i + sub_n; ii++) {
                for(int jj = j; jj < j + sub_n; jj++) {
                    // printf("ii: %d, jj: %d\n", ii, jj);
                    c_reformat[ii * n + jj] = c[count];
                    count++;
                }
            }
        }
    }
#ifdef DEBUG
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", a_tensor[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", b_tensor[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            printf("%f ", c[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
#endif

    gpu_verify(a, b, c_reformat, n, n, n);
    munmap(a, sizeof(double) * n * n);
    munmap(b, sizeof(double) * n * n);
    close(a_fd);
    close(b_fd);

    munmap(a_tensor, sizeof(double) * n * n);
    munmap(b_tensor, sizeof(double) * n * n);
    close(a_tensor_fd);
    close(b_tensor_fd);

    free(c_reformat);
    free(c);
    return 0;
}