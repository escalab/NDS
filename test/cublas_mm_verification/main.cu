#include <stdio.h>
#include <stdlib.h>

#include "cublas_gemm_kernel.cu"
#include "support.cu"
#include "common.cu"

int main (int argc, char *argv[])
{

    Timer timer;
    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000"
      "\n    Usage: ./sgemm-tiled <m>            # All matrices are m x m"
      "\n    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
      "\n");
        exit(0);
    }

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float)*B_sz );
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*C_sz );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol,
        matBrow, matBcol, matArow, matBcol);


    // Launch kernel using msplitm ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    msplitm(matArow, matBcol, matBrow, 1.0f, A_h, matArow, B_h, matBrow, 0.0f, C_h, matBrow);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------
    verify(A_h, B_h, C_h, matArow, matAcol, matBcol);


    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    return 0;

}

