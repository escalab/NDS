#include "hip/hip_runtime.h"
#include "cublasGEMM.h"

__global__ void d2f_kernel(double *din, float *dout, int dsize) {
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

__global__ void d2h_kernel(double *din, half *dout, int dsize) {
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

__global__ void h2f_kernel(half *din, float *dout, int dsize) {
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

float* tensor_blockGemmEx(int x, int y, int z, int sub_m, int sub_n, int sub_k, 
    double *a, double *b, float *c) {
    int i, j, k;
    int cross_row = x * sub_k, cross_col = sub_m * sub_k;
    float alpha = 1.0;
    float beta = 1.0;
    float *a_sub_f, *b_sub_f, *c_sub_f;
    double *a_sub_d, *b_sub_d;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &a_sub_f, sizeof(float) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_f, sizeof(float) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n);

    int dsize = sub_m * sub_n;

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemset(c_sub_f, 0, sub_m * sub_n * sizeof(float));
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                hipMemcpy(a_sub_d, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                hipMemcpy(b_sub_d, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_sub_d, a_sub_f, dsize);
                d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_sub_d, b_sub_f, dsize);
                // async execution (ref: https://forums.developer.nvidia.com/t/async-cublas/2837)
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_f, HIP_R_16F, sub_k, a_sub_f, HIP_R_16F, sub_m, &beta, c_sub_f, HIP_R_32F, sub_m, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
            }
            hipMemcpy((c + i * cross_row + j * cross_col), c_sub_f, sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(a_sub_f);
    hipFree(b_sub_f);
    hipFree(c_sub_f);
    return c;
}

// DON'T USE. Lose precision somewhere.
float* tensor_blockHgemm(int x, int y, int z, int sub_m, int sub_n, int sub_k, 
    double *a, double *b, float *c) {
    int i, j, k;
    int cross_row = x * sub_k, cross_col = sub_m * sub_k;
    half alpha = 1.0;
    half beta = 1.0;
    half *a_sub_h, *b_sub_h, *c_sub_h;
    double *a_sub_d, *b_sub_d;
    float *c_sub_d;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_d, sizeof(float) * sub_m * sub_n);
    hipMalloc((void **) &a_sub_h, sizeof(half) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_h, sizeof(half) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_h, sizeof(half) * sub_m * sub_n);

    int dsize = sub_m * sub_n;

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemset(c_sub_h, 0, sub_m * sub_n * sizeof(half));
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                hipMemcpy(a_sub_d, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                hipMemcpy(b_sub_d, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_sub_d, a_sub_h, dsize);
                d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_sub_d, b_sub_h, dsize);
                hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_h, sub_k, a_sub_h, sub_m, &beta, c_sub_h, sub_m);
            }
            h2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_sub_h, c_sub_d, dsize);
            hipMemcpy((c + i * cross_row + j * cross_col), c_sub_d, sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost);
        }
    }
    
    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    hipFree(a_sub_h);
    hipFree(b_sub_h);
    hipFree(c_sub_h);

    return c;
}

float* tensor_blockSgemm(int x, int y, int z, int sub_m, int sub_n, int sub_k, 
    double *a, double *b, float *c) {
    int i, j, k;
    int cross_row = x * sub_k, cross_col = sub_m * sub_k;
    float alpha = 1.0;
    float beta = 1.0;
    float *a_sub_f, *b_sub_f, *c_sub_f;
    double *a_sub_d, *b_sub_d;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &a_sub_f, sizeof(float) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_f, sizeof(float) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n);

    int dsize = sub_m * sub_n;

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemset(c_sub_f, 0, sub_m * sub_n * sizeof(float));
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                hipMemcpy(a_sub_d, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                hipMemcpy(b_sub_d, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_sub_d, a_sub_f, dsize);
                d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_sub_d, b_sub_f, dsize);
                // async execution (ref: https://forums.developer.nvidia.com/t/async-cublas/2837)
                cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_f, HIP_R_16F, sub_k, a_sub_f, HIP_R_16F, sub_m, &beta, c_sub_f, HIP_R_32F, sub_m);
            }
            hipMemcpy((c + i * cross_row + j * cross_col), c_sub_f, sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(a_sub_f);
    hipFree(b_sub_f);
    hipFree(c_sub_f);
    return c;
}

float* tensor_blockDgemm(int x, int y, int z, int sub_m, int sub_n, int sub_k, 
    double *a, double *b, float *c) {
    int i, j, k;
    int cross_row = x * sub_k, cross_col = sub_m * sub_k;
    double alpha = 1.0;
    double beta = 1.0;
    double *a_sub_d, *b_sub_d, *c_sub_d;
    float *c_sub_f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);


    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_d, sizeof(double) * sub_m * sub_n);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n);

    int dsize = sub_m * sub_n;

    // custom block gemm
    for (i = 0; i < (x / sub_m); i++) {
        for (j = 0; j < (y / sub_n); j++) {
            hipMemset(c_sub_d, 0, sub_m * sub_n * sizeof(double));
            for (k = 0; k < (z / sub_k); k++) {
                // here we can use GPUDirect?
                hipMemcpy(a_sub_d, (a + i * cross_row + k * cross_col), sub_m * sub_k * sizeof(double), hipMemcpyHostToDevice);    
                hipMemcpy(b_sub_d, (b + k * cross_row + j * cross_col), sub_k * sub_n * sizeof(double), hipMemcpyHostToDevice);
                hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, sub_k, a_sub_d, sub_m, &beta, c_sub_d, sub_m);
            }
            d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_sub_d, c_sub_f, dsize);
            hipMemcpy((c + i * cross_row + j * cross_col), c_sub_f, sub_m * sub_n * sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    hipFree(c_sub_f);

    return c;
}

float* sequential_blockSgemm(int x, int y, int z, int sub_m, int sub_n, int sub_k, 
    double *a, double *b, float *c) {
    int i, j, k, ii, kk, i_idx, k_idx;
    float alpha = 1.0;
    float beta = 1.0;
    double *a_sub_d, *b_sub_d;
    float *a_sub_f, *b_sub_f, *c_sub_f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &a_sub_f, sizeof(float) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_f, sizeof(float) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n);

    int dsize = sub_m * sub_n;

    for (i = 0; i < x; i += sub_m) {
        for (j = 0; j < y; j += sub_n) {
            hipMemset(c_sub_f, 0, sub_m * sub_n * sizeof(float));
            for (k = 0; k < z; k += sub_k) {
                for (ii = i, i_idx = 0; ii < (i + sub_m); ii++, i_idx++) {
                    hipMemcpy((a_sub_d + i_idx * sub_n), (a + ii*y + k), sub_k * sizeof(double), hipMemcpyHostToDevice);
                }

                for (kk = k, k_idx = 0; kk < (k + sub_k); kk++, k_idx++) {
                    hipMemcpy((b_sub_d + k_idx * sub_n), (b + kk * y + j), sub_n * sizeof(double), hipMemcpyHostToDevice);
                }

                d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_sub_d, a_sub_f, dsize);
                d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_sub_d, b_sub_f, dsize);
                // hipblasDgemm EXPLANATION ------------------------------------------------
                // the memory layout is different from we know
                // a = [0 1; b = [3 2; 
                //      2 3]      1 0]
                // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
                // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
                // maybe that's because inside GPU it uses column major storage.
                cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_f, HIP_R_16F, sub_k, a_sub_f, HIP_R_16F, sub_m, &beta, c_sub_f, HIP_R_32F, sub_m);
                // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_f, sub_k, a_sub_f, sub_m, &beta, c_sub_f, sub_m);
            }
            for (ii = i, i_idx = 0; ii < (i + sub_n); ii++, i_idx++) {
                hipMemcpy((c + ii * y + j), (c_sub_f + i_idx * sub_n), sub_n * sizeof(float), hipMemcpyDeviceToHost);
            }                
        }
    }  
    
    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(a_sub_f);
    hipFree(b_sub_f);
    hipFree(c_sub_f);

    return c;
}

float* sequential_blockDgemm(int x, int y, int z, int sub_m, int sub_n, int sub_k, 
    double *a, double *b, float *c) {
    int i, j, k, ii, kk, i_idx, k_idx;
    double alpha = 1.0;
    double beta = 1.0;
    double *a_sub_d, *b_sub_d, *c_sub_d;
    float *c_sub_f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_sub_d, sizeof(double) * sub_m * sub_k);
    hipMalloc((void **) &b_sub_d, sizeof(double) * sub_k * sub_n);
    hipMalloc((void **) &c_sub_d, sizeof(double) * sub_m * sub_n);
    hipMalloc((void **) &c_sub_f, sizeof(float) * sub_m * sub_n);

    int dsize = sub_m * sub_n;

    for (i = 0; i < x; i += sub_m) {
        for (j = 0; j < y; j += sub_n) {
            hipMemset(c_sub_d, 0, sub_m * sub_n * sizeof(double));
            for (k = 0; k < z; k += sub_k) {
                for (ii = i, i_idx = 0; ii < (i + sub_m); ii++, i_idx++) {
                    hipMemcpy((a_sub_d + i_idx * sub_n), (a + ii*y + k), sub_k * sizeof(double), hipMemcpyHostToDevice);
                }

                for (kk = k, k_idx = 0; kk < (k + sub_k); kk++, k_idx++) {
                    hipMemcpy((b_sub_d + k_idx * sub_n), (b + kk * y + j), sub_n * sizeof(double), hipMemcpyHostToDevice);
                }
                // hipblasDgemm EXPLANATION ------------------------------------------------
                // the memory layout is different from we know
                // a = [0 1; b = [3 2; 
                //      2 3]      1 0]
                // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
                // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
                // maybe that's because inside GPU it uses column major storage.
                hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, sub_k, &alpha, b_sub_d, sub_k, a_sub_d, sub_m, &beta, c_sub_d, sub_m);
            }
            d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_sub_d, c_sub_f, dsize);
            for (ii = i, i_idx = 0; ii < (i + sub_n); ii++, i_idx++) {
                hipMemcpy((c + ii * y + j), (c_sub_f + i_idx * sub_n), sub_n * sizeof(float), hipMemcpyDeviceToHost);
            }                
        }
    }  
    
    hipblasDestroy(handle);

    hipFree(a_sub_d);
    hipFree(b_sub_d);
    hipFree(c_sub_d);
    hipFree(c_sub_f);

    return c;
}

// DON'T USE. Lose precision somewhere.
float* wholeMatrixHgemm(int m, int n, int k, const double *a, const double *b, float *c) {
    half alpha = 1.0;
    half beta = 0.0;
    double *a_d, *b_d;
    half *a_h, *b_h, *c_h;
    float *c_f;
    int dsize = m * n;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_d, sizeof(double) * m * k);
    hipMalloc((void **) &b_d, sizeof(double) * k * n);
    hipMalloc((void **) &a_h, sizeof(half) * m * k);
    hipMalloc((void **) &b_h, sizeof(half) * k * n);
    hipMalloc((void **) &c_h, sizeof(half) * k * n);
    hipMalloc((void **) &c_f, sizeof(float) * m * n);

    hipMemcpy(a_d, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(double) * k * n, hipMemcpyHostToDevice);

    d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_d, a_h, dsize);
    d2h_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_d, b_h, dsize);

    hipFree(a_d);
    hipFree(b_d);
    // hipblasDgemm EXPLANATION ------------------------------------------------
    // the memory layout is different from we know
    // a = [0 1; b = [3 2; 
    //      2 3]      1 0]
    // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
    // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
    // maybe that's because inside GPU it uses column major storage.
    hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_h, k, a_h, m, &beta, c_h, m);
    hipFree(a_h);
    hipFree(b_h);
    h2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_h, c_f, dsize);
    hipMemcpy(c, c_f, sizeof(float) * m * n, hipMemcpyDeviceToHost);
    
    hipblasDestroy(handle);
    hipFree(c_h);
    hipFree(c_f);
    return c;
}

float* wholeMatrixSgemm(int m, int n, int k, const double *a, const double *b, float *c) {
    float alpha = 1.0;
    float beta = 0.0;
    double *a_d, *b_d;
    float *a_f, *b_f, *c_f;
    int dsize = m * n;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    hipMalloc((void **) &a_d, sizeof(double) * m * k);
    hipMalloc((void **) &b_d, sizeof(double) * k * n);
    hipMalloc((void **) &a_f, sizeof(float) * m * k);
    hipMalloc((void **) &b_f, sizeof(float) * k * n);
    hipMalloc((void **) &c_f, sizeof(float) * m * n);

    hipMemcpy(a_d, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(double) * k * n, hipMemcpyHostToDevice);

    d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(a_d, a_f, dsize);
    d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(b_d, b_f, dsize);

    hipFree(a_d);
    hipFree(b_d);
    // hipblasDgemm EXPLANATION ------------------------------------------------
    // the memory layout is different from we know
    // a = [0 1; b = [3 2; 
    //      2 3]      1 0]
    // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
    // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
    // maybe that's because inside GPU it uses column major storage.
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_f, HIP_R_16F, k, a_f, HIP_R_16F, m, &beta, c_f, HIP_R_32F, m);
    cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_f, HIP_R_16F, k, a_f, HIP_R_16F, m, &beta, c_f, HIP_R_32F, m);

    hipFree(a_f);
    hipFree(b_f);

    hipMemcpy(c, c_f, sizeof(float) * m * n, hipMemcpyDeviceToHost);
    
    hipblasDestroy(handle);

    hipFree(c_f);
    return c;
}

float* wholeMatrixDgemm(int m, int n, int k, const double *a, const double *b, float *c) {
    double alpha = 1.0;
    double beta = 0.0;
    double *a_d, *b_d, *c_d;
    float *c_f;
    int dsize = m * n;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **) &a_d, sizeof(double) * m * k);
    hipMalloc((void **) &b_d, sizeof(double) * k * n);
    hipMalloc((void **) &c_d, sizeof(double) * m * n);

    hipMemcpy(a_d, a, sizeof(double) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(double) * k * n, hipMemcpyHostToDevice);

    // hipblasDgemm EXPLANATION ------------------------------------------------
    // the memory layout is different from we know
    // a = [0 1; b = [3 2; 
    //      2 3]      1 0]
    // if use a_d then b_d, c[0][0] will be a[0, 0] * b[0, 0] + a[1, 0] * b[0, 1] = 4
    // with b_d then a_d, c[0][0] will be a[0, 0] * b[0, 0] + a[0, 1] * b[1, 0] = 1
    // maybe that's because inside GPU it uses column major storage.
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, b_d, k, a_d, m, &beta, c_d, m);
    hipFree(a_d);
    hipFree(b_d);

    hipMalloc((void **) &c_f, sizeof(float) * m * n);
    d2f_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(c_d, c_f, dsize);
    hipMemcpy(c, c_f, sizeof(float) * m * n, hipMemcpyDeviceToHost);
    
    hipblasDestroy(handle);

    hipFree(c_d);
    hipFree(c_f);
    return c;
}