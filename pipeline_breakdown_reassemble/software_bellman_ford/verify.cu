#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_runtime.h>

// for mmap
#include <sys/mman.h>
#include <fcntl.h>
#include <unistd.h>

extern "C" {
    #include "rdma.h"
    #include "timing.h"
    #include "fifo.h"
}

#define INF INT_MAX
#define THREADS_PER_BLOCK 1024

#define HUGEPAGE_SZ (4UL * 1024UL * 1024UL * 1024UL)
#define M 65536UL
#define SUB_M 4096UL
#define AGGREGATED_SZ (M * SUB_M * 8UL)

#define IO_QUEUE_SZ (HUGEPAGE_SZ / AGGREGATED_SZ)
// #define IO_QUEUE_SZ 1UL

#define NITERS 16UL // pre-check from another implementation

void print_config(struct config_t config);

struct fetch_conf {
    struct resources *res;
    uint64_t m, sub_m;
    int64_t *graph_d;
    char *hugepage_addr;
    struct fifo *sending_queue;
    struct fifo *complete_queue;
    struct timing_info *col_fetch_timing;
    struct timing_info *copy_in_timing;
};

struct request_conf {
    struct resources *res;
    uint64_t id;
    uint64_t sub_m;
};

struct fifo_entry {
    int64_t *graph_d;
};

void colstripe_reshape(double *dst, double *src, size_t m, size_t sub_m, size_t granularity) {
    uint64_t chunk, i, row, col;
    uint64_t dsize = m * sub_m;
    uint64_t multiplier = sub_m / granularity;
    double *dst_ptr, *src_ptr = src;
    for (chunk = 0; chunk < (dsize / granularity / granularity); chunk++) {
        row = chunk / multiplier;
        col = chunk % multiplier;
        dst_ptr = dst + row * sub_m * granularity + col * granularity; 
        for (i = 0; i < granularity; i++) {
            memcpy(dst_ptr, src_ptr, sizeof(double) * granularity);
            dst_ptr += sub_m;
            src_ptr += granularity;
        }
    }
}

__global__ void bellman_ford_one_iter(size_t n, size_t sub_n, size_t st, int64_t *d_subgraph, int64_t *d_dist, bool *d_has_next){
	size_t v = blockDim.x * blockIdx.x + threadIdx.x;
	size_t u;
	int64_t weight, new_dist;
	int64_t *node;
	
	if (v > sub_n) {
		return;
	}
	
	node = d_subgraph + v;
	v = v + st;
	for (u = 0; u < n; u++){
		weight = node[u * sub_n]; // row is src, col is dst
		if (weight > 0) {
			new_dist = d_dist[u] + weight;
			if(new_dist < d_dist[v]){
				d_dist[v] = new_dist;
				*d_has_next = true;
			}
		}
	}
}

int cudaMemcpyFromMmap(struct fetch_conf *conf, uint64_t id, uint64_t st, uint64_t en, uint64_t size, uint64_t op, uint64_t which,
    char *dst, const char *src, const size_t length, struct timing_info *fetch_timing) {
    struct response *res = NULL;

    timing_info_push_start(fetch_timing);
    sock_write_request(conf->res->req_sock, id, st, en, size, op, which);
    sock_read_data(conf->res->req_sock);

    res = sock_read_offset(conf->res->sock);
    if (res == NULL) {
        fprintf(stderr, "sync error before RDMA ops\n");
        return 1;
    }

    // if (res->id == 0) {
    //     printf("fetching row [%lu:%lu]\n", res->x, res->y);
    // } else {
    //     printf("fetching col [%lu:%lu]\n", res->x, res->y);
    // }
    // printf("offset: %lu\n", res->offset);

    timing_info_push_end(fetch_timing);

    timing_info_push_start(conf->copy_in_timing);
    hipMemcpy(dst, src + res->offset, length, hipMemcpyHostToDevice);
    timing_info_push_end(conf->copy_in_timing);

    free(res);
    if (sock_write_data(conf->res->sock)) { /* just send a dummy char back and forth */
        fprintf(stderr, "sync error before RDMA ops\n");
        return 1;
    }
    return 0;
}

void *request_thread(void *args) {
    struct request_conf *conf = (struct request_conf *) args;
    uint64_t st;

    for (st = 0; st < M / SUB_M; st++) {
        sock_write_request(conf->res->req_sock, conf->id, st, st+1, conf->sub_m, 3, 0);
        sock_read_data(conf->res->req_sock);
    }
    return NULL;
}

int nds_bfs(struct resources *res, uint64_t id, uint64_t num_of_vertices, uint64_t num_of_subvertices) {
    size_t stripe_size, st, i;
    int64_t *graph_d;

    // result
    int64_t *dist, *d_dist;
    bool *d_has_next, h_has_next;

    size_t total_iteration;

	uint64_t iter = 0;

    struct timing_info *col_fetch_timing;
    struct timing_info *reshape_timing;    
    struct timing_info *copy_in_timing;
    struct timing_info *kernel_timing;
    struct timing_info *copy_out_timing;    
    
    struct fetch_conf f_conf;

    struct timeval h_start, h_end;
    long duration;

    struct response *resp = NULL;
    double *reshaped_data = (double *) calloc(M * SUB_M, sizeof(double));

    // initialization
    total_iteration = NITERS * (num_of_vertices / num_of_subvertices);

    col_fetch_timing = timing_info_new(total_iteration);
    if (col_fetch_timing == NULL) {
        printf("cannot create col_fetch_timing\n");
        return -1;
    }
    
    reshape_timing = timing_info_new(total_iteration);
    if (reshape_timing == NULL) {
        printf("cannot create reshape_timing\n");
        return -1;
    }

    copy_in_timing = timing_info_new(total_iteration * 2);
    if (copy_in_timing == NULL) {
        printf("cannot create copy_in_timing\n");
        return -1;
    }

    kernel_timing = timing_info_new(total_iteration);
    if (kernel_timing == NULL) {
        printf("cannot create kernel_timing\n");
        return -1;
    }

    copy_out_timing = timing_info_new(total_iteration);
    if (copy_out_timing == NULL) {
        printf("cannot create copy_out_timing\n");
        return -1;
    }

    // subgraph initialization
    stripe_size = num_of_vertices * num_of_subvertices * sizeof(int64_t);
    hipMalloc((void **) &graph_d, stripe_size);

    // Bellman-Ford initialization
	hipMalloc(&d_dist, sizeof(int64_t) * num_of_vertices);
	hipMalloc(&d_has_next, sizeof(bool));

    bool has_negative_cycle = false;
    
    dist = (int64_t *) calloc(sizeof(int64_t), num_of_vertices);
	for(i = 0 ; i < num_of_vertices; i++){
		dist[i] = INF;
	}

	dist[0] = 0;
    hipMemcpy(d_dist, dist, sizeof(int64_t) * num_of_vertices, hipMemcpyHostToDevice);

    // create thread here
    f_conf.res = res;
    f_conf.m = num_of_vertices;
    f_conf.sub_m = num_of_subvertices;
    f_conf.graph_d = graph_d;
    f_conf.hugepage_addr = res->buf;
    f_conf.col_fetch_timing = col_fetch_timing;
    f_conf.copy_in_timing = copy_in_timing;

    timing_info_set_starting_time(col_fetch_timing);
    timing_info_set_starting_time(reshape_timing);
    timing_info_set_starting_time(copy_in_timing);
    timing_info_set_starting_time(kernel_timing);
    timing_info_set_starting_time(copy_out_timing);

    gettimeofday(&h_start, NULL);
    // blockGEMM
    do {
        printf("iter: %lu\n", iter);
        //if no thread changes this value then the loop stops
		h_has_next = false;
        hipMemset(d_has_next, 0, sizeof(bool));
        
        for (st = 0; st < num_of_vertices / num_of_subvertices; st++) {
            timing_info_push_start(col_fetch_timing);
            sock_write_request(res->req_sock, id, st, st+1, SUB_M, 3, 0);
            sock_read_data(res->req_sock);
        
            resp = sock_read_offset(res->sock);
            if (resp == NULL) {
                fprintf(stderr, "sync error before RDMA ops\n");
                return -1;
            }

            timing_info_push_end(col_fetch_timing);

            timing_info_push_start(reshape_timing);
            colstripe_reshape(reshaped_data, (double *) (res->buf + resp->offset), M, SUB_M, 256UL);
            if (sock_write_data(res->sock)) { /* just send a dummy char back and forth */
                fprintf(stderr, "sync error before RDMA ops\n");
                return -1;
            }
            free(resp);
            timing_info_push_end(reshape_timing);

            timing_info_push_start(copy_in_timing);
            hipMemcpy(graph_d, reshaped_data, M * SUB_M * sizeof(double), hipMemcpyHostToDevice);
            timing_info_push_end(copy_in_timing);

            timing_info_push_start(kernel_timing);
			bellman_ford_one_iter<<<(num_of_subvertices+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(num_of_vertices, num_of_subvertices, st * num_of_subvertices, graph_d, d_dist, d_has_next);
            timing_info_push_end(kernel_timing);
        }
        
        timing_info_push_start(copy_out_timing);
		hipMemcpy(&h_has_next, d_has_next, sizeof(bool), hipMemcpyDeviceToHost);
        timing_info_push_end(copy_out_timing);

        iter++;
        if (iter >= num_of_vertices - 1){
			has_negative_cycle = true;
			break;
		}
    } while (h_has_next);

    // send a signal to tell storage backend the iteration is done.
    sock_write_request(res->req_sock, -1, st, st+1, SUB_M, 0, 0);
    sock_read_data(res->req_sock);
    
    if (!has_negative_cycle){
        hipMemcpy(dist, d_dist, sizeof(int64_t) * num_of_vertices, hipMemcpyDeviceToHost);
	} 
    gettimeofday(&h_end, NULL);
    duration = ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);
    
    printf("Bellman-Ford End-to-end duration: %f ms\n", (float) duration / 1000);    
	printf("Kernel Executed %lu times\n", iter);

    printf("Col fetch time: %f ms\n", (float) timing_info_duration(col_fetch_timing) / 1000);
    printf("Reshape time: %f ms\n", (float) timing_info_duration(reshape_timing) / 1000);
    printf("Copy in time: %f ms\n", (float) timing_info_duration(copy_in_timing) / 1000);
    printf("Kernel time: %f ms\n", (float) timing_info_duration(kernel_timing) / 1000);
    printf("copy out time: %f ms\n", (float) timing_info_duration(copy_out_timing) / 1000);
    
    struct timestamps *tss = NULL;
    FILE *fptr;
    tss = timing_info_get_timestamps(col_fetch_timing);
    fptr = fopen("col_fetch_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(col_fetch_timing);

    tss = timing_info_get_timestamps(reshape_timing);
    fptr = fopen("reshape_timing_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(reshape_timing);

    tss = timing_info_get_timestamps(copy_in_timing);
    fptr = fopen("copy_in_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(copy_in_timing);

    tss = timing_info_get_timestamps(kernel_timing);
    fptr = fopen("kernel_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(kernel_timing);

    tss = timing_info_get_timestamps(copy_out_timing);
    fptr = fopen("copy_out_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(copy_out_timing);

    
    fptr = fopen("log.txt", "w");
    if (!has_negative_cycle){
        for (i = 0; i < num_of_vertices; i++) {
			if (dist[i] > INF) {
				dist[i] = INF;
			}
			fprintf(fptr, "%lu %lu\n", i, dist[i]);
        }
	} else {
		fprintf(fptr, "FOUND NEGATIVE CYCLE!\n");
	}
    fclose(fptr);

	// cleanup
    free(dist);
    free(reshaped_data);
    hipFree(graph_d);
	hipFree(d_dist);
    hipFree(d_has_next);
    return 0;
}

/******************************************************************************
 * Function: print_config
 *
 * Input
 * none
 *
 * Output
 * none
 *
 * Returns
 * none
 *
 * Description
 * Print out config information
 ******************************************************************************/
void print_config(struct config_t config) {
    fprintf(stdout, " ------------------------------------------------\n");
    fprintf(stdout, " Device name : \"%s\"\n", config.dev_name);
    fprintf(stdout, " IB port : %u\n", config.ib_port);
    if (config.server_name)
        fprintf(stdout, " IP : %s\n", config.server_name);
    fprintf(stdout, " TCP port : %u\n", config.tcp_port);
    if (config.gid_idx >= 0)
        fprintf(stdout, " GID index : %u\n", config.gid_idx);
    fprintf(stdout, " ------------------------------------------------\n\n");
}

int main(int argc, char *argv[]) {
    int rc = 0;
    uint64_t matrix_id, n, sub_n;

    int hugepage_fd;
    char *hugepage_addr;

    // RDMA
    struct resources res;
    struct config_t config = {
        "mlx4_0",  /* dev_name */
        NULL,  /* server_name */
        19875, /* tcp_port */
        1,     /* ib_port */
        0     /* gid_idx */
    };

    // default the iteration is 4 times
    if (argc < 5) {
        printf("usage: %s <matrix_id> <# of vertices> <# of subvertices> <port>\n", argv[0]);
        exit(1);
    } 
    matrix_id = (uint64_t) atoll(argv[1]);
    n = (uint64_t) atoll(argv[2]);
    sub_n = (uint64_t) atoll(argv[3]);
    config.tcp_port = atoi(argv[4]);

    /* print the used parameters for info*/
    print_config(config);
    
    printf("mapping hugepage\n");
    hugepage_fd = open("/dev/hugepages/tensorstore", O_RDWR, 0755);
    if (hugepage_fd < 0) {
        perror("open");
        exit(1);
    }

    hugepage_addr = (char *) mmap(0, BUF_SIZE, PROT_READ | PROT_WRITE, MAP_SHARED, hugepage_fd, 0);
    if (hugepage_addr==MAP_FAILED) {
        perror("mmap");
        exit(1);
    }

    res.buf = hugepage_addr;
    memset(hugepage_addr, 0, BUF_SIZE);

    printf("hugepage starting address is: %p\n", hugepage_addr);
    printf("socket connection\n");
    rc = make_two_tcp_connection(&res, &config);
    if (rc < 0) {
        perror("sock connect");
        exit(1);
    }

    printf("calculating the result of pagerank\n");
    rc = nds_bfs(&res, matrix_id, n, sub_n);
    
    close(res.sock);
    close(res.req_sock);
    munmap(hugepage_addr, BUF_SIZE);
    close(hugepage_fd);
    return rc;
}
