#include "hip/hip_runtime.h"
extern "C" {
    #include "rdma.h"
    #include "timing.h"
    #include "fifo.h"
}

#define RANDOMRESETPROB 0.15
#define NUM_THREADS 1024

#define HUGEPAGE_SZ (4UL * 1024UL * 1024UL * 1024UL)
#define M 65536UL
#define SUB_M 4096UL
#define AGGREGATED_SZ (M * SUB_M * 8UL)

#define IO_QUEUE_SZ (HUGEPAGE_SZ / AGGREGATED_SZ / 2UL)
// #define IO_QUEUE_SZ 1UL

#define NITERS 4UL

void print_config(struct config_t config);

struct fetch_conf {
    struct resources *res;
    uint64_t m, sub_m;
    int64_t *outedges, *inedges;
    char *hugepage_addr;
    struct fifo *sending_queue;
    struct fifo *complete_queue;
    struct timing_info *row_fetch_timing;
    struct timing_info *col_fetch_timing;
    struct timing_info *copy_in_timing;
};

struct request_conf {
    struct resources *res;
    uint64_t id;
    uint64_t sub_m;
};

struct fifo_entry {
    int64_t *outedges, *inedges;
};

void colstripe_reshape(double *dst, double *src, size_t m, size_t sub_m, size_t granularity) {
    uint64_t chunk, i, row, col;
    uint64_t dsize = m * sub_m;
    uint64_t multiplier = sub_m / granularity;
    double *dst_ptr, *src_ptr = src;
    for (chunk = 0; chunk < (dsize / granularity / granularity); chunk++) {
        row = chunk / multiplier;
        col = chunk % multiplier;
        dst_ptr = dst + row * sub_m * granularity + col * granularity; 
        for (i = 0; i < granularity; i++) {
            memcpy(dst_ptr, src_ptr, sizeof(double) * granularity);
            dst_ptr += sub_m;
            src_ptr += granularity;
        }
    }
}

void rowstripe_reshape(double *dst, double *src, size_t m, size_t sub_m, size_t granularity) {
    uint64_t chunk, i, row, col;
    uint64_t dsize = m * sub_m;
    uint64_t multiplier = m / granularity;
    double *dst_ptr, *src_ptr = src;
    for (chunk = 0; chunk < (dsize / granularity / granularity); chunk++) {
        row = chunk / multiplier;
        col = chunk % multiplier;
        dst_ptr = dst + row * m * granularity + col * granularity; 
        for (i = 0; i < granularity; i++) {
            memcpy(dst_ptr, src_ptr, sizeof(double) * granularity);
            dst_ptr += m;
            src_ptr += granularity;
        }
    }
}

__global__ void pagerank_update(double* prev_pr, double* curr_pr, double *vertices, size_t st, int64_t* inedges, int64_t *outedges, int m, int sub_m, int iter, int niters) {
    // v.outc is num_outedges()
    // needs: v.num_inedges(), v.inedge(), v.id(), v.outc, v.set_data
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (id >= sub_m) {
        return;
    }

    int64_t *outedge = outedges + id * m;
    int64_t *inedge = inedges + id;
    int i, outc = 0;
    double sum = 0;

    id = st + id;
    for (i = 0; i < m; i++) {
        if (i != id && outedge[i] != 0) {
            outc++;
        }
    }

    // first iteration
    if (iter > 0) {
        for (i = 0; i < m; i++) {
            // we don't consider self-loop
            if (inedge[i * sub_m] && i != id) {
                sum += prev_pr[i];
            }
        }
        if (outc > 0) {
            curr_pr[id] = (RANDOMRESETPROB + (1 - RANDOMRESETPROB) * sum) / (double) outc;
        } else {
            curr_pr[id] = (RANDOMRESETPROB + (1 - RANDOMRESETPROB) * sum);
        }
    } else if (iter == 0) {
        if (outc > 0) {
            curr_pr[id] = 1.0f / (double) outc;
        }
    }

    // for the last iteration
    if (iter == niters - 1) {
        if (outc > 0) {
            vertices[id] = curr_pr[id] * (double) outc;
        } else {
            vertices[id] = curr_pr[id];
        }
    }
}

int cudaMemcpyFromMmap(struct fetch_conf *conf, uint64_t id, uint64_t st, uint64_t en, uint64_t size, uint64_t op, uint64_t which,
    char *dst, const char *src, const size_t length, struct timing_info *fetch_timing) {
    struct response *res = NULL;

    timing_info_push_start(fetch_timing);
    sock_write_request(conf->res->req_sock, id, st, en, size, op, which);
    sock_read_data(conf->res->req_sock);

    res = sock_read_offset(conf->res->sock);
    if (res == NULL) {
        fprintf(stderr, "sync error before RDMA ops\n");
        return 1;
    }

    // if (res->id == 0) {
    //     printf("fetching row [%lu:%lu]\n", res->x, res->y);
    // } else {
    //     printf("fetching col [%lu:%lu]\n", res->x, res->y);
    // }
    // printf("offset: %lu\n", res->offset);

    timing_info_push_end(fetch_timing);

    timing_info_push_start(conf->copy_in_timing);
    hipMemcpy(dst, src + res->offset, length, hipMemcpyHostToDevice);
    timing_info_push_end(conf->copy_in_timing);

    free(res);
    if (sock_write_data(conf->res->sock)) { /* just send a dummy char back and forth */
        fprintf(stderr, "sync error before RDMA ops\n");
        return 1;
    }
    return 0;
}

void *request_thread(void *args) {
    struct request_conf *conf = (struct request_conf *) args;
    uint64_t i, st;

    for (i = 0; i < NITERS; i++) {
        for (st = 0; st < M / SUB_M; st++) {
            sock_write_request(conf->res->req_sock, conf->id, st, st+1, SUB_M, 2, 0);
            sock_read_data(conf->res->req_sock);

            sock_write_request(conf->res->req_sock, conf->id, st, st+1, SUB_M, 3, 1);
            sock_read_data(conf->res->req_sock);
        }
    }
    sock_write_request(conf->res->req_sock, -1, st, st+1, SUB_M, 0, 0);
    sock_read_data(conf->res->req_sock);
    return NULL;
}

int nds_pagerank(struct resources *res, uint64_t id, uint64_t m, uint64_t sub_m) {
    size_t i, st;    
    int64_t *outedges, *inedges;

    // result
    double *vertices;
    double *prev_pr_d, *curr_pr_d, *vertices_d;
    
    size_t total_iteration;
    uint64_t stripe_size;

    struct timing_info *row_fetch_timing;
    struct timing_info *row_reshape_timing;    
    struct timing_info *col_fetch_timing;
    struct timing_info *col_reshape_timing;    
    struct timing_info *copy_in_timing;
    struct timing_info *pagerank_timing;
    struct timing_info *copy_out_timing;    
    
    struct fetch_conf f_conf;

    struct timeval h_start, h_end;
    long duration;

    struct response *resp = NULL;
    double *reshaped_data = (double *) calloc(M * SUB_M, sizeof(double));

    // initialization
    total_iteration = NITERS * (m / sub_m);

    row_fetch_timing = timing_info_new(total_iteration);
    if (row_fetch_timing == NULL) {
        printf("cannot create row_fetch_timing\n");
        return -1;
    }

    row_reshape_timing = timing_info_new(total_iteration);
    if (row_reshape_timing == NULL) {
        printf("cannot create row_reshape_timing\n");
        return -1;
    }

    col_fetch_timing = timing_info_new(total_iteration);
    if (col_fetch_timing == NULL) {
        printf("cannot create col_fetch_timing\n");
        return -1;
    }

    col_reshape_timing = timing_info_new(total_iteration);
    if (col_reshape_timing == NULL) {
        printf("cannot create col_reshape_timing\n");
        return -1;
    }

    copy_in_timing = timing_info_new(total_iteration * 2);
    if (copy_in_timing == NULL) {
        printf("cannot create copy_in_timing\n");
        return -1;
    }

    pagerank_timing = timing_info_new(total_iteration);
    if (pagerank_timing == NULL) {
        printf("cannot create pagerank_timing\n");
        return -1;
    }

    copy_out_timing = timing_info_new(total_iteration);
    if (copy_out_timing == NULL) {
        printf("cannot create copy_out_timing\n");
        return -1;
    }

    // subgraph initialization
    stripe_size = m * sub_m * sizeof(int64_t);
    hipMalloc((void **) &outedges, stripe_size);
    hipMalloc((void **) &inedges, stripe_size);

    // PR initialization
    vertices = (double *) malloc(sizeof(double) * m);
    for (i = 0; i < m; i++) {
        vertices[i] = RANDOMRESETPROB;
    }

    // malloc
    hipMalloc((void **) &vertices_d, sizeof(double) * m);
    hipMalloc((void **) &prev_pr_d, sizeof(double) * m);
    hipMalloc((void **) &curr_pr_d, sizeof(double) * m);

    // setup memory
    hipMemset(vertices_d, 0, sizeof(double) * m);
    hipMemcpy(prev_pr_d, vertices, sizeof(double) * m, hipMemcpyHostToDevice);
    hipMemset(curr_pr_d, 0, sizeof(double) * m);
    memset(vertices, 0, sizeof(double) * m);

    // create thread here
    f_conf.res = res;
    f_conf.m = m;
    f_conf.sub_m = sub_m;
    f_conf.outedges = outedges;
    f_conf.inedges = inedges;
    f_conf.hugepage_addr = res->buf;
    f_conf.row_fetch_timing = row_fetch_timing;
    f_conf.col_fetch_timing = col_fetch_timing;
    f_conf.copy_in_timing = copy_in_timing;

    timing_info_set_starting_time(row_fetch_timing);
    timing_info_set_starting_time(row_reshape_timing);
    timing_info_set_starting_time(col_fetch_timing);
    timing_info_set_starting_time(col_reshape_timing);
    timing_info_set_starting_time(copy_in_timing);
    timing_info_set_starting_time(pagerank_timing);
    timing_info_set_starting_time(copy_out_timing);

    gettimeofday(&h_start, NULL);
    // blockGEMM
    for (i = 0; i < NITERS; i++) {
        printf("iter: %lu\n", i);
        for (st = 0; st < m / sub_m; st++) {
            printf("st: %lu\n", st * sub_m);
            // outedges
            timing_info_push_start(row_fetch_timing);
            sock_write_request(res->req_sock, id, st, st+1, SUB_M, 2, 0);
            sock_read_data(res->req_sock);
        
            resp = sock_read_offset(res->sock);
            if (resp == NULL) {
                fprintf(stderr, "sync error before RDMA ops\n");
                return -1;
            }

            timing_info_push_end(row_fetch_timing);

            timing_info_push_start(row_reshape_timing);
            rowstripe_reshape(reshaped_data, (double *) (res->buf + resp->offset), M, SUB_M, 256UL);
            if (sock_write_data(res->sock)) { /* just send a dummy char back and forth */
                fprintf(stderr, "sync error before RDMA ops\n");
                return -1;
            }
            free(resp);
            timing_info_push_end(row_reshape_timing);

            timing_info_push_start(copy_in_timing);
            hipMemcpy(outedges, reshaped_data, M * SUB_M * sizeof(double), hipMemcpyHostToDevice);
            timing_info_push_end(copy_in_timing);

            // inedges
            timing_info_push_start(col_fetch_timing);
            sock_write_request(res->req_sock, id, st, st+1, SUB_M, 3, 0);
            sock_read_data(res->req_sock);
        
            resp = sock_read_offset(res->sock);
            if (resp == NULL) {
                fprintf(stderr, "sync error before RDMA ops\n");
                return -1;
            }

            timing_info_push_end(col_fetch_timing);

            timing_info_push_start(col_reshape_timing);
            colstripe_reshape(reshaped_data, (double *) (res->buf + resp->offset), M, SUB_M, 256UL);
            if (sock_write_data(res->sock)) { /* just send a dummy char back and forth */
                fprintf(stderr, "sync error before RDMA ops\n");
                return -1;
            }
            free(resp);
            timing_info_push_end(col_reshape_timing);

            timing_info_push_start(copy_in_timing);
            hipMemcpy(inedges, reshaped_data, M * SUB_M * sizeof(double), hipMemcpyHostToDevice);
            timing_info_push_end(copy_in_timing);
    
            timing_info_push_start(pagerank_timing);
            pagerank_update<<<(sub_m+NUM_THREADS-1)/NUM_THREADS, NUM_THREADS>>>(prev_pr_d, curr_pr_d, vertices_d, st * sub_m, inedges, outedges, m, sub_m, i, NITERS);
            timing_info_push_end(pagerank_timing);
        }

        timing_info_push_start(copy_out_timing);
        hipMemcpy(prev_pr_d, curr_pr_d, sizeof(double) * m, hipMemcpyDeviceToDevice);
        timing_info_push_end(copy_out_timing);
    }
    hipMemcpy(vertices, vertices_d, sizeof(double) * m, hipMemcpyDeviceToHost);

    sock_write_request(res->req_sock, -1, st, st+1, SUB_M, 0, 0);
    sock_read_data(res->req_sock);

    gettimeofday(&h_end, NULL);
    duration = ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);
    printf("GEMM duration: %f ms\n", (float) duration / 1000);    

    printf("Row fetch time: %f ms\n", (float) timing_info_duration(row_fetch_timing) / 1000);
    printf("Row reshape time: %f ms\n", (float) timing_info_duration(row_reshape_timing) / 1000);

    printf("Col fetch time: %f ms\n", (float) timing_info_duration(col_fetch_timing) / 1000);
    printf("Col reshape time: %f ms\n", (float) timing_info_duration(col_reshape_timing) / 1000);

    printf("Copy in time: %f ms\n", (float) timing_info_duration(copy_in_timing) / 1000);
    printf("GEMM time: %f ms\n", (float) timing_info_duration(pagerank_timing) / 1000);
    printf("copy out time: %f ms\n", (float) timing_info_duration(copy_out_timing) / 1000);
    
    struct timestamps *tss = NULL;
    FILE *fptr;
    tss = timing_info_get_timestamps(row_fetch_timing);
    fptr = fopen("row_fetch_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(row_fetch_timing);

    tss = timing_info_get_timestamps(row_reshape_timing);
    fptr = fopen("row_reshape_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(row_reshape_timing);

    tss = timing_info_get_timestamps(col_fetch_timing);
    fptr = fopen("col_fetch_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(col_fetch_timing);

    tss = timing_info_get_timestamps(col_reshape_timing);
    fptr = fopen("col_reshape_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(col_reshape_timing);

    tss = timing_info_get_timestamps(copy_in_timing);
    fptr = fopen("copy_in_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(copy_in_timing);

    tss = timing_info_get_timestamps(pagerank_timing);
    fptr = fopen("gemm_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(pagerank_timing);

    tss = timing_info_get_timestamps(copy_out_timing);
    fptr = fopen("copy_out_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(copy_out_timing);

    fptr = fopen("log.txt", "w");
    for (i = 0; i < m; i++) {
        fprintf(fptr, "%lu %f\n", i, vertices[i]);
    }
    fclose(fptr);
    free(reshaped_data);
    hipFree(outedges);
    hipFree(inedges);
    hipFree(vertices_d);
    hipFree(prev_pr_d);
    hipFree(curr_pr_d);
    free(vertices);
    return 0;
}

/******************************************************************************
 * Function: print_config
 *
 * Input
 * none
 *
 * Output
 * none
 *
 * Returns
 * none
 *
 * Description
 * Print out config information
 ******************************************************************************/
void print_config(struct config_t config) {
    fprintf(stdout, " ------------------------------------------------\n");
    fprintf(stdout, " Device name : \"%s\"\n", config.dev_name);
    fprintf(stdout, " IB port : %u\n", config.ib_port);
    if (config.server_name)
        fprintf(stdout, " IP : %s\n", config.server_name);
    fprintf(stdout, " TCP port : %u\n", config.tcp_port);
    if (config.gid_idx >= 0)
        fprintf(stdout, " GID index : %u\n", config.gid_idx);
    fprintf(stdout, " ------------------------------------------------\n\n");
}

int main(int argc, char *argv[]) {
    int rc = 0;
    uint64_t matrix_id, n, sub_n;

    int hugepage_fd;
    char *hugepage_addr;

    // RDMA
    struct resources res;
    struct config_t config = {
        "mlx4_0",  /* dev_name */
        NULL,  /* server_name */
        19875, /* tcp_port */
        1,     /* ib_port */
        0     /* gid_idx */
    };

    // default the iteration is 4 times
    if (argc < 5) {
        printf("usage: %s <matrix_id> <# of vertices> <# of subvertices> <port>\n", argv[0]);
        exit(1);
    } 
    matrix_id = (uint64_t) atoll(argv[1]);
    n = (uint64_t) atoll(argv[2]);
    sub_n = (uint64_t) atoll(argv[3]);
    config.tcp_port = atoi(argv[4]);

    /* print the used parameters for info*/
    print_config(config);
    
    printf("mapping hugepage\n");
    hugepage_fd = open("/dev/hugepages/tensorstore", O_RDWR, 0755);
    if (hugepage_fd < 0) {
        perror("open");
        exit(1);
    }

    hugepage_addr = (char *) mmap(0, BUF_SIZE, PROT_READ | PROT_WRITE, MAP_SHARED, hugepage_fd, 0);
    if (hugepage_addr==MAP_FAILED) {
        perror("mmap");
        exit(1);
    }

    res.buf = hugepage_addr;
    memset(hugepage_addr, 0, BUF_SIZE);

    printf("hugepage starting address is: %p\n", hugepage_addr);
    printf("socket connection\n");
    rc = make_two_tcp_connection(&res, &config);
    if (rc < 0) {
        perror("sock connect");
        exit(1);
    }

    printf("calculating the result of pagerank\n");
    rc = nds_pagerank(&res, matrix_id, n, sub_n);
    
    close(res.sock);
    close(res.req_sock);
    munmap(hugepage_addr, BUF_SIZE);
    close(hugepage_fd);
    return rc;
}
