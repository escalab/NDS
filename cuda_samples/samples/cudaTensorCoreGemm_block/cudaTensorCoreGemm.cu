#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// CUDA sample demonstrating a GEMM computation using the Warp Matrix Multiply
// and Accumulate API introduced in CUDA 9.

// In this program, the compute_gemm kernel computes the result of a matrix
// multiplication and addition: D = alpha * A * B + beta * C. The dimensions of
// both C and D matrices are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x
// K_GLOBAL (row-major), the B matrix is K_GLOBAL x N_GLOBAL (column-major). In
// that kernel, each CTA computes one 128 x 128 tile of the resulting matrix per
// iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 128 x 128 tile to compute.
// Each CTA consists of eight warps. For the 128 x 128 tile, each warp computes
// eight 16 x 16 subtiles, organized in a 2 x 4 two-dimensional array. Warps
// compute the 16 x 16 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and
// accumulating the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 128 x 128 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments
//   from shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B
// matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the
//   A and B data from shared memory, thus reducing the number of data copies
//   from global memory.
// - The portions of the A and B matrices are stored in shared memory with an
// additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_HALF macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each
// warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory
//   contents to global memory, again avoiding redundant random global memory
//   accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register
// utilization,
//   but carefully enough to avoid local memory use.

#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Externally configurable parameters.

#ifndef CPU_DEBUG
// Set this to 1 to verify the correctness of the GPU-computed matrix.
#define CPU_DEBUG 0
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 1
#endif

// GPU configuration.
#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 32
#define N 32
#define K 32

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that are 16 * 16 * 8 * 8 * 2 = 32 Kb each
// (i.e. two 8x8 arrays of tiles of 16x16 half-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the
// performance would be severely impacted. So we choose to reduce the chunk size
// in half, i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B
// matrix in shared memory to minimize possible bank conflicts. Before
// performing the nvcuda::wmma::mma_sync operation, the warp must load the
// matrix data using the nvcuda::wmma::load_matrix_sync operation. Although the
// memory access pattern is not specified for that function, each lane in the
// warp can read one or multiple matrix elements from different matrix rows or
// columns. For shared memory, such access can result in bank conflicts if
// different rows / columns of the matrix map to the same bank. By shifting each
// row and column by a few bytes, we make sure that they map to different banks,
// thus reducing the number of possible bank conflicts. The number of 8 two-byte
// "half" elements is chosen as the minimum possible shift because we must keep
// each row and column 128-bit aligned, as required by
// nvcuda::wmma::load_matrix_sync.
#define SKEW_HALF 8

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;

__host__ void init_host_matrices(half *a, half *b, float *c, int M_GLOBAL, int N_GLOBAL, int K_GLOBAL) {
  for (int i = 0; i < M_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      a[i * K_GLOBAL + j] = (half)(rand() % 3);
    }
  }

  for (int i = 0; i < N_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      b[i * K_GLOBAL + j] = (half)(rand() % 3);
    }
  }

  for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
    c[t] = static_cast<float>(rand() % 3);
  }
}

__host__ void init_host_matrices(half *a, int M_GLOBAL, int N_GLOBAL) {
  for (int i = 0; i < M_GLOBAL; i++) {
    for (int j = 0; j < N_GLOBAL; j++) {
      a[i * N_GLOBAL + j] = (half) (rand() % 3);
    }
  }
}

__host__ void init_host_matrices(double *a, int M_GLOBAL, int N_GLOBAL) {
  for (int i = 0; i < M_GLOBAL; i++) {
    for (int j = 0; j < N_GLOBAL; j++) {
      a[i * N_GLOBAL + j] = ((double) rand() / RAND_MAX);
    }
  }
}

__host__ void init_host_matrices(float *c, int M_GLOBAL, int N_GLOBAL) {
  for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
    c[t] = static_cast<float>(rand() % 3);
  }
}


__global__ void half_conversion_kernel(double *din, half *dout, int dsize) {
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < dsize)
	{
		dout[idx] = din[idx];
	}
}

// Calculate AB with NVIDIA TensorCores
// Kernel executed by 1 Warp (32 Threads)
__global__ void tensorOp(half *a, half *b, float *c) {
  // Tile using a 2D grid
  // int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // [0, 1]
  // int warpN = (blockIdx.y * blockDim.y + threadIdx.y); // [0, 1]
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
  int cRow = warpM * WMMA_M;
  int cCol = warpN * WMMA_N;
  // printf("warpM: %d, warpN: %d\n", warpM, warpN);
  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
  nvcuda::wmma::load_matrix_sync(c_frag, c + cRow * N + cCol, N, wmma::mem_row_major);

  // Loop over k
  for (int i = 0; i < K; i += WMMA_K) {
    int aRow = warpM * WMMA_M;
    int aCol = i;
    int bCol = warpN * WMMA_N;
    int bRow = i;

    // Bounds checking
    if (aRow < M && aCol < K && bRow < K && bCol < N) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aCol + aRow * M, M);
      wmma::load_matrix_sync(b_frag, b + bCol + bRow * K, K);

      // 4. Perform the matrix multiplication
      nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
  }
  // 5. Store the result from fragment to global
  nvcuda::wmma::store_matrix_sync(c + cRow * N + cCol, c_frag, N, nvcuda::wmma::mem_row_major);
}

// too complicated and use too many macros to ignore this first
/**
__global__ void compute_gemm(const half *A, const half *B, const float *C,
                             float *D, float alpha, float beta) {
  extern __shared__ half shmem[][CHUNK_K * K + SKEW_HALF];

  // Warp and lane identification.
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  // Offset in shared memory from which the B matrix is stored.
  const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

  // This pointer is used to access the C and D matrix tiles this warp computes.
  float *shmem_warp_tile_ptr = (float *)&shmem[0][0] +
                               (warpId / 2) * SHMEM_STRIDE * K * 2 +
                               (warpId % 2) * SHMEM_OFFSET;

  // This pointer is used to stream the C and D matrices block-wide tile to and
  // from shared memory.
  float *shmem_warp_stream_ptr =
      (float *)&shmem[0][0] + warpId * SHMEM_STRIDE * K;

  // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
  // each tile computation. Technically this is not generally correct (may
  // result in a loss of precision). Zero still needs to be specially handled
  // though.
  beta /= alpha;

  // Each CTA slides along the 128 x 128 tiles from the top left corner of the
  // matrix to the right and down, and selects the next tile to compute. Once
  // there's no such tile, all warps in this CTA exit.
  for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
    const unsigned int block_tile_i =
        ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
    const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

    // Stop when there are no more D matrix tiles to compute in this CTA.
    if (block_tile_i >= M_TILES) {
      break;
    }

    // This warp's pointer to the C matrix data to copy memory from to shared
    // memory.
    const size_t gmem_idx =
        (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
    const float *src_gmem_warp_stream_ptr = &C[gmem_idx];

    // Stream multiple C tiles to shared memory.
#pragma unroll
    for (int i = 0; i < K; i++) {
      typedef int4 copy_t;

      *((copy_t *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) =
          *((copy_t *)(src_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) +
            laneId);
    }

    __syncthreads();

    // These fragments will accumulate the result of A and B matrix fragment
    // multiplications along the K_GLOBAL dimension.
    wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES]
                                                       [WARP_ROW_TILES];

    // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
        const float *tile_ptr =
            shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

        wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
      }
    }

    __syncthreads();

    // Scale the C matrix.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
        for (int t = 0; t < c[i][j].num_elements; t++) {
          c[i][j].x[t] *= beta;
        }
      }
    }

    // Select what warp copies what matrix to shared memory.
    // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
    const half *warp_ptr = (warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] +
                                           M * K_GLOBAL * (warpId % 4) * 2)
                                        : (&B[block_tile_j * N * K_GLOBAL] +
                                           N * K_GLOBAL * (warpId % 4) * 2);

    // Go through the global K dimension by a fixed step at a time.
#pragma unroll
    for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
      // Copy slices of the A and B matrices to shared memory.
      // The first half of the warps in the CTA copy the A matrix, the rest copy
      // the B matrix.
      size_t shmem_idx =
          warpId < (WARPS_PER_BLOCK / 2)
              ? (M * (warpId % (WARPS_PER_BLOCK / 2)) * 2)
              : (N * (warpId % (WARPS_PER_BLOCK / 2)) * 2 + shmem_idx_b_off);

      // First half of the warp copies the first row / column of the matrix,
      // the second half of the warp copies the next.
      int4 *lane_ptr = (int4 *)(warp_ptr + tile_k * K +
                                (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL) +
                       (laneId % CHUNK_COPY_LINE_LANES);

      // Shift the second half of the warp to the next row / column in the
      // shared memory.
      shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
      for (int i = 0; i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP) * 2;
           i++) {
        // Copy 16 bytes at once in each lane.
        *((int4 *)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) =
            *lane_ptr;

        // Advance the global memory pointer and the shared memory index.
        lane_ptr =
            (int4 *)((half *)lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP);
        shmem_idx += CHUNK_COPY_LINES_PER_WARP;
      }

      __syncthreads();

      // Compute a grid of C matrix tiles in each warp.
#pragma unroll
      for (int k_step = 0; k_step < CHUNK_K; k_step++) {
        wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major>
            a[WARP_COL_TILES];
        wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major>
            b[WARP_ROW_TILES];

#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
          size_t shmem_idx_a = (warpId / 2) * M * 2 + (i * M);
          const half *tile_ptr = &shmem[shmem_idx_a][k_step * K];

          wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_HALF);

#pragma unroll
          for (int j = 0; j < WARP_ROW_TILES; j++) {
            if (i == 0) {
              // Load the B matrix fragment once, because it is going to be
              // reused against the other A matrix fragments.
              size_t shmem_idx_b = shmem_idx_b_off +
                                   (WARP_ROW_TILES * N) * (warpId % 2) +
                                   (j * N);
              const half *tile_ptr = &shmem[shmem_idx_b][k_step * K];

              wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_HALF);
            }

            wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
          }
        }
      }

      __syncthreads();
    }

      // Store the D fragments to shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
        // Uniform, point-wise transformations of ALL fragment elements by ALL
        // threads in the warp are well-defined even though element indices
        // within fragment storage are not defined.
        for (int t = 0; t < c[i][j].num_elements; t++) c[i][j].x[t] *= alpha;

        float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

        wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
      }
    }

    __syncthreads();

    // Now that shared memory contains all the D tiles, stream them to global
    // memory.
    float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
    for (int i = 0; i < K; i++) {
      *((int4 *)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
          *((int4 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
    }

    __syncthreads();
  }
}
*/

// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is a less performant version of the compute_gemm kernel. It is
// designed for
//       demonstration purposes only to show the CUDA WMMA API use without
//       relying on availability of the shared memory.
__global__ void simple_wmma_gemm(half *a, half *b, float *c, float *d, int m_ld,
                                 int n_ld, int k_ld, float alpha, float beta) {
  // Leading dimensions. Packed with no transpositions.
  int lda = m_ld;
  int ldb = k_ld;
  int ldc = n_ld;

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  // Loop over k
  for (int i = 0; i < k_ld; i += WMMA_K) {
    int aCol = i;
    int aRow = warpM * WMMA_M;

    int bCol = i;
    int bRow = warpN * WMMA_N;

    // Bounds checking
    if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bCol + bRow * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cCol = warpN * WMMA_N;
  int cRow = warpM * WMMA_M;

  if (cRow < m_ld && cCol < n_ld) {
    wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc,
                           wmma::mem_row_major);

    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc,
                            wmma::mem_row_major);
  }
}

__host__ void matMultiplyOnHost(half *A, half *B, float *C, float alpha,
                                float beta, int numARows, int numAColumns,
                                int numBRows, int numBColumns, int numCRows,
                                int numCColumns) {
  for (int i = 0; i < numCRows; i++) {
    for (int j = 0; j < numCColumns; j++) {
      float temp = 0.0;

      for (int k = 0; k < numAColumns; k++) {
        temp += (float)A[i * numAColumns + k] * (float)B[j * numBRows + k];
      }

      C[i * numCColumns + j] = temp * alpha + beta * C[i * numCColumns + j];
    }
  }
}

__host__ void matMultiplyOnHost(double *A, double *B, float *C, float alpha,
                                float beta, int numARows, int numAColumns,
                                int numBRows, int numBColumns, int numCRows,
                                int numCColumns) {
  for (int i = 0; i < numCRows; i++) {
    for (int j = 0; j < numCColumns; j++) {
      for (int k = 0; k < numAColumns; k++) {
        // beware of row-major or col-major store
        // C[i * numCColumns + j] += (float)A[i * numAColumns + k] * (float)B[j * numBRows + k];
        C[i * numCColumns + j] += (float)A[i * numAColumns + k] * (float)B[k * numBColumns + j];
      }

      // C[i * numCColumns + j] = temp * alpha + beta * C[i * numCColumns + j];
    }
  }
}

__host__ void blockMatMultiplyOnHost(double *A, double *B, float *C, float alpha,
  float beta, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
  int i, j, k, ii, jj, kk;
  for (i = 0; i < numCRows; i += M) {
    for (j = 0; j < numCColumns; j += N) {
      for (k = 0; k < numAColumns; k += K) {
        for (ii = i; ii < (i + M); ii++) {
          for (jj = j; jj < (j + N); jj++) {
            for (kk = k; kk < (k + K); kk++) {
              C[ii * numCColumns + jj] += A[ii * numAColumns + kk] * B[kk * numBColumns + jj];
            }
          }
        }
      }
      // C[i * numCColumns + j] = temp * alpha + beta * C[i * numCColumns + j];
    }
  }
}


int main(int argc, char **argv) {
  const float alpha = 1.0f;
  const float beta = 0.0f;
  float milliseconds = 0;
  int M_TILES, N_TILES, K_TILES, M_GLOBAL, N_GLOBAL, K_GLOBAL;
  hipEvent_t start, stop;
  FILE *fp;

  int i, j, k;
  size_t dsize = M * K;
  dim3 gridDim;
  dim3 blockDim;
  printf("Initializing...\n");


#if CPU_DEBUG
  if (argc < 6) {
    printf("usage: %s <A_matrix_path> <m_global> <n_global> <k_global> <answer_path>\n", argv[0]);
    exit(1);
  }
#else
  if (argc < 5) {
    printf("usage: %s <A_matrix_path> <m_global> <n_global> <k_global>\n", argv[0]);
    exit(1);
  }
#endif

  // GEMM configuration.
  fp = fopen(argv[1], "rb");
  M_GLOBAL = atoi(argv[2]);
  N_GLOBAL = atoi(argv[3]);
  K_GLOBAL = atoi(argv[4]);

  M_TILES = M_GLOBAL / M / WMMA_M;
  N_TILES = N_GLOBAL / N / WMMA_N;
  K_TILES = K_GLOBAL / K / WMMA_K;
  int dev = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
  // Tensor cores require a GPU of Volta (SM7X) architecture or higher.
  if (deviceProp.major < 7) {
    printf(
        "cudaTensorCoreGemm requires SM 7.0 or higher to use Tensor "
        "Cores.  Exiting...\n");
    exit(EXIT_WAIVED);
  }
  
  printf("The warp size is %d.\n", deviceProp.warpSize);

  // blockDim.x must be a multple of warpSize
  // 128x4 means we have 16 warps and a block computes a 64x64 output tile
  // for RTX 2080, we have 1024 threads per block.
  blockDim.y = N / WMMA_N; // 32 / 16 = 2
  blockDim.x = deviceProp.warpSize * blockDim.y; // 32 * 2 = 64
  gridDim.x = (M_GLOBAL + (M * blockDim.x / deviceProp.warpSize - 1)) /
              (M * blockDim.x / deviceProp.warpSize);
  gridDim.y = (N_GLOBAL + N * blockDim.y - 1) / (N * blockDim.y);
  printf("gridDim.x=%d, gridDim.y=%d, blockDim.x=%d, blockDim.y=%d\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  printf("M: %d (%d x %d x %d)\n", M_GLOBAL, M, WMMA_M, M_TILES);
  printf("N: %d (%d x %d x %d)\n", N_GLOBAL, N, WMMA_N, N_TILES);
  printf("K: %d (%d x %d x %d)\n", K_GLOBAL, K, WMMA_K, K_TILES);

  double *A_h = NULL;
  double *B_h = NULL;
  float *C_h = NULL;
  A_h = (double *)malloc(sizeof(double) * M_GLOBAL * K_GLOBAL);
  B_h = (double *)malloc(sizeof(double) * K_GLOBAL * N_GLOBAL);
  C_h = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);

#if CPU_DEBUG
  double *A_submatrix_h = NULL;
  double *B_submatrix_h = NULL;
  float *C_submatrix_h = NULL;
  float *result_host = NULL;
  float *answer = NULL;

  A_submatrix_h = (double *)malloc(sizeof(double) * M * K);
  B_submatrix_h = (double *)malloc(sizeof(double) * K * N);
  C_submatrix_h = (float *)malloc(sizeof(float) * M * N);
  answer = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  result_host = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  memset(result_host, 0, sizeof(float) * M_GLOBAL * N_GLOBAL);
#endif

  int count;
  count = fread(A_h, sizeof(double), M_GLOBAL * K_GLOBAL, fp);
  if (count != M_GLOBAL * K_GLOBAL) {
    printf("read num of element mismatched! count: %d, matrix_size: %d\n",count, M_GLOBAL * K_GLOBAL);
  }

  fseek(fp, 0, SEEK_SET);

  count = fread(B_h, sizeof(double), K_GLOBAL * N_GLOBAL, fp);
  if (count != K_GLOBAL * N_GLOBAL) {
    printf("read num of element mismatched! count: %d, matrix_size: %d\n",count, K_GLOBAL * N_GLOBAL);
  }
  
  // init_host_matrices(B_h, N_GLOBAL, K_GLOBAL);
  // init_host_matrices(C_h, M_GLOBAL, N_GLOBAL);
  memset(C_h, 0, sizeof(float) * M_GLOBAL * N_GLOBAL);

  // printf("A = \n");
  // for (i = 0; i < M_GLOBAL; i++) {
  //   for (j = 0; j < K_GLOBAL; j++) {
  //     printf("%f ", A_h[i*K_GLOBAL+j]);
  //   }
  //   printf("\n");
  // }
  // printf("\n");

  // printf("B = \n");
  // for (i = 0; i < K_GLOBAL; i++) {
  //   for (j = 0; j < N_GLOBAL; j++) {
  //     printf("%f ", B_h[i*N_GLOBAL+j]);
  //   }
  //   printf("\n");
  // }
  // printf("\n");


  // printf("C = \n");
  // for (i = 0; i < M_GLOBAL; i++) {
  //   for (j = 0; j < N_GLOBAL; j++) {
  //     printf("%f ", C_h[i*N_GLOBAL+j]);
  //   }
  //   printf("\n");
  // }

  printf("\n");

  double *A_double = NULL;
  half *A = NULL;
  double *B_double = NULL;
  half *B = NULL;
  float *C = NULL;
  float *D = NULL;

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&A_double),
                             sizeof(double) * M * K));  
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&A),
                             sizeof(half) * M * K));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&B_double),
                             sizeof(double) * M * K));  
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&B),
                             sizeof(half) * N * K));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&C),
                             sizeof(float) * M * N));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&D),
                             sizeof(float) * M * N));

  assert(((unsigned long long)A) % 128 == 0);
  assert(((unsigned long long)B) % 128 == 0);
  assert(((unsigned long long)C) % 128 == 0);
  assert(((unsigned long long)D) % 128 == 0);


  printf("Preparing data for GPU...\n");
  checkCudaErrors(hipMemset(C, 0, sizeof(float) * M * N));

  checkCudaErrors(hipEventRecord(start));
  // custom block gemm
  int cross_row = M_GLOBAL * K, cross_col = M * K;
  for (i = 0; i < (M_GLOBAL / M); i++) {
    for (j = 0; j < (N_GLOBAL / N); j++) {
        checkCudaErrors(hipMemcpy(C, (C_h + i * cross_row + j * cross_col), M * N * sizeof(float), hipMemcpyHostToDevice));
        for (k = 0; k < (K_GLOBAL / K); k++) {
            // fill the block
            // printf("%p\n", A_h + i * cross_row + k * cross_col);
            checkCudaErrors(hipMemcpy(B_double, (B_h + k * cross_row + j * cross_col), K * N * sizeof(double), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(A_double, (A_h + i * cross_row + k * cross_col), M * K * sizeof(double), hipMemcpyHostToDevice));    
            half_conversion_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(A_double, A, dsize);
            half_conversion_kernel<<<(dsize+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(B_double, B, dsize);
            tensorOp<<<1, blockDim>>>(A, B, C);
        }
        checkCudaErrors(hipMemcpy((C_h + i * cross_row + j * cross_col), C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    }
  }

  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Time: %f ms\n", milliseconds);
  printf("TFLOPS: %.2f\n", static_cast<double>((static_cast<double>(M_GLOBAL) *
                                                N_GLOBAL * K_GLOBAL * 2) /
                                               (milliseconds / 1000.)) / 1e12);
#if CPU_DEBUG
  printf("Verifying correctness of the computations...\n");
  // matMultiplyOnHost(A_h, B_h, result_host, alpha, beta, M_GLOBAL, K_GLOBAL,
  //                   K_GLOBAL, N_GLOBAL, M_GLOBAL, N_GLOBAL);
  checkCudaErrors(hipEventRecord(start));
  // blockMatMultiplyOnHost(A_h, B_h, result_host, alpha, beta, M_GLOBAL, K_GLOBAL,
  //   K_GLOBAL, N_GLOBAL, M_GLOBAL, N_GLOBAL);
  
  // custom block gemm
  cross_row = M_GLOBAL * K;
  cross_col = K * K;

  for (i = 0; i < (M_GLOBAL / K); i++) {
    for (j = 0; j < (N_GLOBAL / N); j++) {
        memcpy(C_submatrix_h, (result_host + i * cross_row + j * cross_col), K * N * sizeof(float));
        for (k = 0; k < (K_GLOBAL / K); k++) {
            // fill the block
            memcpy(A_submatrix_h, (A_h + i * cross_row + k * cross_col), K * K * sizeof(double));
            memcpy(B_submatrix_h, (B_h + k * cross_row + j * cross_col), K * N * sizeof(double));
            matMultiplyOnHost(A_submatrix_h, B_submatrix_h, C_submatrix_h, alpha, beta, K, K, K, N, K, N);
        }
        memcpy((result_host + i * cross_row + j * cross_col), C_submatrix_h, K * N * sizeof(float));
    }
  }
  
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Time: %f ms\n", milliseconds);

  fp = fopen(argv[5], "rb");
  fread(answer, sizeof(float), M_GLOBAL * N_GLOBAL, fp);
  fclose(fp);

  count = 0;
  for (int i = 0; i < M_GLOBAL * N_GLOBAL; i++) {
    if (fabs(answer[i] - result_host[i]) > 0.1f) {
      printf("mismatch i=%d answer=%f result_host=%f\n", i, answer[i], result_host[i]);
      count++;
    }
  }

  if (count == 0) {
    printf("host computation test passed\n");
  }

  count = 0;
  for (int i = 0; i < M_GLOBAL * N_GLOBAL; i++) {
    if (fabs(C_h[i] - result_host[i]) > 0.1f) {
      printf("mismatch i=%d C_h=%f result_host=%f\n", i, C_h[i], result_host[i]);
      count++;
    }             
  }

  if (count == 0) {
    printf("TensorOP test passed\n");
  }

  free(answer);

  free(result_host);
  free(A_submatrix_h);
  free(B_submatrix_h);
  free(C_submatrix_h);
#endif

  free(A_h);
  free(B_h);
  free(C_h);
  checkCudaErrors(hipFree(reinterpret_cast<void *>(A_double)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(A)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(B_double)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(B)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(C)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(D)));
  return 0;
}
