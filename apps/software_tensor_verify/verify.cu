extern "C" {
    #include "rdma.h"
    #include "timing.h"
    #include "fifo.h"
}

#include <fcntl.h>
#include <unistd.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#define M 1024UL
#define N M
#define K M

#define LDB1 M
#define LDB2 LDB1*N
#define LDC1 N

#define SUB_M 512UL

#define HUGEPAGE_SZ (4UL * 1024UL * 1024UL * 1024UL)
#define AGGREGATED_SZ (SUB_M * SUB_M * SUB_M * 8UL)

// #define IO_QUEUE_SZ (HUGEPAGE_SZ / AGGREGATED_SZ / 2UL)
#define IO_QUEUE_SZ 1UL

#define NITERS 4UL

void print_config(struct config_t config);

struct fetch_conf {
    struct resources *res;
    uint64_t m, sub_m;
    double *sub_B;
    char *hugepage_addr;
    struct fifo *sending_queue;
    struct fifo *complete_queue;
    struct timing_info *fetch_timing;
    struct timing_info *copy_in_timing;
};

struct request_conf {
    struct resources *res;
    uint64_t id;
    uint64_t sub_m;
};

struct fifo_entry {
    double *sub_B;
};

int memcpyFromMmap(struct fetch_conf *conf, char *dst, const char *src, const size_t length, struct timing_info *fetch_timing) {
    struct response *res = NULL;

    timing_info_push_start(fetch_timing);
    res = sock_read_offset(conf->res->sock);
    if (res == NULL) {
        fprintf(stderr, "sync error before RDMA ops\n");
        return 1;
    }

    // if (res->id == 0) {
    //     printf("fetching row [%lu:%lu]\n", res->x, res->y);
    // } else {
    //     printf("fetching col [%lu:%lu]\n", res->x, res->y);
    // }
    timing_info_push_end(fetch_timing);

    timing_info_push_start(conf->copy_in_timing);

    memcpy(dst, src + res->offset, length);
    timing_info_push_end(conf->copy_in_timing);

    free(res);
    if (sock_write_data(conf->res->sock)) { /* just send a dummy char back and forth */
        fprintf(stderr, "sync error before RDMA ops\n");
        return 1;
    }
    return 0;
}

void *fetch_thread(void *args) {
    struct fetch_conf *conf = (struct fetch_conf *) args;
    uint64_t n, m, k;
    uint64_t dsize = SUB_M * SUB_M * SUB_M;
    double *ptr_a;
    struct fifo_entry *entry = NULL;
    uint64_t count = 0;

    for (n = 0; n < N / SUB_M; n++) {
        for (m = 0; m < M / SUB_M; m++) {
            for (k = 0; k < K / SUB_M; k++) {
                entry = (struct fifo_entry *) fifo_pop(conf->complete_queue);
                ptr_a = conf->sub_B + dsize * (count % IO_QUEUE_SZ);

                memcpyFromMmap(conf, (char *) ptr_a, (char *) conf->hugepage_addr, dsize * sizeof(double), conf->fetch_timing);
                count++;

                entry->sub_B = ptr_a;
                fifo_push(conf->sending_queue, entry);
            }
        }
    }
    return NULL;
}

void *request_thread(void *args) {
    struct request_conf *conf = (struct request_conf *) args;
    uint64_t n, m, k;
    for (n = 0; n < N / SUB_M; n++) {
        for (m = 0; m < M / SUB_M; m++) {
            for (k = 0; k < K / SUB_M; k++) {
                sock_write_request(conf->res->req_sock, conf->id, n, m, k, 1, 0);
                sock_read_data(conf->res->req_sock);
            }
        }
    }

    sock_write_request(conf->res->req_sock, -1, n, m, k, 1, 0);
    sock_read_data(conf->res->req_sock);
    return NULL;
}

void generate_data(double *array, uint64_t size) {
    uint64_t i;
    for (i = 0; i < size; i++) {
        // array[i] = (((double) rand())/RAND_MAX - 0.5)*100;
        array[i] = i;        
    }
}

uint64_t verify(const double *answer_C, const double *C, uint64_t size) {
    uint64_t i, error = 0;
    for (i = 0; i < size; i++) {
        if (answer_C[i] != C[i]) {
            printf("index %lu error, answer_C: %f, C: %f\n", i, answer_C[i], C[i]);
            error++;
            return error;
        }
    }
    return error;
}

void reassemble_block_tensor_from_seq(const double *seq_matrix, double *result_matrix, size_t n, size_t sub_n, size_t i, size_t j, size_t k) {
    size_t ii, jj, kk, a, b, c;
    for (ii = i, a = 0; ii < i+sub_n; ii++, a++) {
        for (jj = j, b = 0; jj < j+sub_n; jj++, b++) {
            // printf("i=%lu, j=%lu, k=%lu, result_offset=%lu, seq_offset=%lu\n", ii, jj, k, b * sub_n + a * sub_n * sub_n, (ii * n * n * sizeof(double) + jj * n * sizeof(double) + k * sizeof(double)));
            for (kk = k, c = 0; kk < k+sub_n; kk++, c++) {
                result_matrix[c + b * sub_n + a * sub_n * sub_n] = seq_matrix[ii * n * n + jj * n + kk];
            }
        }
    }
}

int nds_tensor_verify(struct resources *res, uint64_t id, uint64_t size, uint64_t sub_size, const double *verify_matrix) {
    double *sub_B, *verify_sub_B;
    size_t i, n, m, k, nn, mm, kk, a, b, c;
    size_t error = 0;
    
    size_t total_iteration;

    struct fifo *sending_queue;
    struct fifo *complete_queue; 
    struct fifo_entry *entries = (struct fifo_entry *) calloc(IO_QUEUE_SZ, sizeof(struct fifo_entry));
    struct fifo_entry *entry = NULL;

    struct timing_info *queue_timing;
    struct timing_info *fetch_timing;
    struct timing_info *copy_in_timing;
    struct timing_info *ttv_timing;
    struct timing_info *copy_out_timing;    
    
    pthread_t f_thread_id; 
    struct fetch_conf f_conf;

    pthread_t r_thread_id; 
    struct request_conf r_conf;

    struct timeval h_start, h_end;
    long duration;

    // initialization
    total_iteration = (M / SUB_M) * (M / SUB_M) * (M / SUB_M);
    queue_timing = timing_info_new(total_iteration);
    if (queue_timing == NULL) {
        printf("cannot create queue_timing\n");
        return -1;
    }

    fetch_timing = timing_info_new(total_iteration);
    if (fetch_timing == NULL) {
        printf("cannot create fetch_timing\n");
        return -1;
    }

    copy_in_timing = timing_info_new(total_iteration * 2);
    if (copy_in_timing == NULL) {
        printf("cannot create copy_in_timing\n");
        return -1;
    }

    ttv_timing = timing_info_new(total_iteration);
    if (ttv_timing == NULL) {
        printf("cannot create ttv_timing\n");
        return -1;
    }

    copy_out_timing = timing_info_new(total_iteration);
    if (copy_out_timing == NULL) {
        printf("cannot create copy_out_timing\n");
        return -1;
    }

    // it causes problem if size == 1
    sending_queue = fifo_new(IO_QUEUE_SZ * 2);
	if (sending_queue == NULL) {
        printf("cannot create sending_queue\n");
        return -1;
    }
    
    complete_queue = fifo_new(IO_QUEUE_SZ * 2);
	if (complete_queue == NULL) {
        printf("cannot create complete_queue\n");
        return -1;
    }
    
    for (i = 0; i < IO_QUEUE_SZ; i++) {
        fifo_push(complete_queue, entries + i);
    }

    sub_B = (double *) malloc(SUB_M * SUB_M * SUB_M * sizeof(double));
    verify_sub_B = (double *) malloc(SUB_M * SUB_M * SUB_M * sizeof(double));

    // M * N has to be < 1024
    dim3 grid((SUB_M+32)/32, (SUB_M+32)/32);
    dim3 block(32, 32);

    r_conf.res = res;
    r_conf.id = id;
    r_conf.sub_m = SUB_M;
    pthread_create(&r_thread_id, NULL, request_thread, &r_conf); 

    // create thread here
    f_conf.res = res;
    f_conf.m = size;
    f_conf.sub_m = sub_size;
    f_conf.sub_B = sub_B;
    f_conf.hugepage_addr = res->buf;
    f_conf.sending_queue = sending_queue;
    f_conf.complete_queue = complete_queue;
    f_conf.fetch_timing = fetch_timing;
    f_conf.copy_in_timing = copy_in_timing;

    timing_info_set_starting_time(queue_timing);
    timing_info_set_starting_time(fetch_timing);
    timing_info_set_starting_time(copy_in_timing);
    timing_info_set_starting_time(ttv_timing);
    timing_info_set_starting_time(copy_out_timing);
	pthread_create(&f_thread_id, NULL, fetch_thread, &f_conf); 

    gettimeofday(&h_start, NULL);
    // blockGEMM
    for (n = 0; n < N; n+=SUB_M) {
        for (m = 0; m < M; m+=SUB_M) { 
            for (k = 0; k < K; k+=SUB_M) {
                // memcpy?

                timing_info_push_start(queue_timing);
                entry = (struct fifo_entry *) fifo_pop(sending_queue);
                timing_info_push_end(queue_timing);
                timing_info_push_start(ttv_timing);
                reassemble_block_tensor_from_seq(verify_matrix, verify_sub_B, M, SUB_M, n, m, k);
                error += verify(verify_sub_B, entry->sub_B, SUB_M * SUB_M * SUB_M);
                fifo_push(complete_queue, entry);
                timing_info_push_end(ttv_timing);
            }
            // assign C
            timing_info_push_start(copy_out_timing);
            timing_info_push_end(copy_out_timing);
        }
    }
            
    gettimeofday(&h_end, NULL);
    duration = ((h_end.tv_sec - h_start.tv_sec) * 1000000) + (h_end.tv_usec - h_start.tv_usec);
    printf("TTV duration: %f ms\n", (float) duration / 1000);    

    printf("Row fetch time: %f ms\n", (float) timing_info_duration(fetch_timing) / 1000);
    printf("Copy in time: %f ms\n", (float) timing_info_duration(copy_in_timing) / 1000);
    printf("sending_queue waiting time: %f ms\n", (float) timing_info_duration(queue_timing) / 1000);
    printf("Kernel time: %f ms\n", (float) timing_info_duration(ttv_timing) / 1000);
    printf("copy out time: %f ms\n", (float) timing_info_duration(copy_out_timing) / 1000);
    
    if (error == 0) {
        printf("test passed!\n");
    }
    pthread_join(r_thread_id, NULL); 
    pthread_join(f_thread_id, NULL); 

    struct timestamps *tss = NULL;
    FILE *fptr;
    tss = timing_info_get_timestamps(fetch_timing);
    fptr = fopen("row_fetch_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(fetch_timing);

    tss = timing_info_get_timestamps(copy_in_timing);
    fptr = fopen("copy_in_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(copy_in_timing);
    
    tss = timing_info_get_timestamps(queue_timing);
    fptr = fopen("queue_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(queue_timing);

    tss = timing_info_get_timestamps(ttv_timing);
    fptr = fopen("gemm_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(ttv_timing);

    tss = timing_info_get_timestamps(copy_out_timing);
    fptr = fopen("copy_out_ts.bin", "wb");
    fwrite(&tss->count, sizeof(uint64_t), 1, fptr);
    fwrite(tss->timestamps, sizeof(uint64_t), tss->count * 2, fptr);
    fclose(fptr);
    timing_info_free_timestamps(tss);    
    timing_info_free(copy_out_timing);
    
    free(sub_B);
    free(verify_sub_B);
    fifo_free(sending_queue);
    fifo_free(complete_queue);
    free(entries);
    return 0;
}

/******************************************************************************
 * Function: print_config
 *
 * Input
 * none
 *
 * Output
 * none
 *
 * Returns
 * none
 *
 * Description
 * Print out config information
 ******************************************************************************/
void print_config(struct config_t config) {
    fprintf(stdout, " ------------------------------------------------\n");
    fprintf(stdout, " Device name : \"%s\"\n", config.dev_name);
    fprintf(stdout, " IB port : %u\n", config.ib_port);
    if (config.server_name)
        fprintf(stdout, " IP : %s\n", config.server_name);
    fprintf(stdout, " TCP port : %u\n", config.tcp_port);
    if (config.gid_idx >= 0)
        fprintf(stdout, " GID index : %u\n", config.gid_idx);
    fprintf(stdout, " ------------------------------------------------\n\n");
}

int main(int argc, char *argv[]) {
    int rc = 0;
    uint64_t matrix_id, n, sub_n;

    int verify_fd, hugepage_fd;
    char *hugepage_addr;
    double *verify_matrix;

    // RDMA
    struct resources res;
    struct config_t config = {
        "mlx4_0",  /* dev_name */
        NULL,  /* server_name */
        19875, /* tcp_port */
        1,     /* ib_port */
        0     /* gid_idx */
    };

    // default the iteration is 4 times
    if (argc < 4) {
        printf("usage: %s <matrix_id> <verify_matrix_path> <port>\n", argv[0]);
        exit(1);
    } 
    
    matrix_id = (uint64_t) atoll(argv[1]);
    config.tcp_port = atoi(argv[3]);

    /* print the used parameters for info*/
    print_config(config);
    
    printf("mapping hugepage\n");
    hugepage_fd = open("/dev/hugepages/tensorstore", O_RDWR, 0755);
    if (hugepage_fd < 0) {
        perror("open");
        exit(1);
    }

    hugepage_addr = (char *) mmap(0, BUF_SIZE, PROT_READ | PROT_WRITE, MAP_SHARED, hugepage_fd, 0);
    if (hugepage_addr==MAP_FAILED) {
        perror("mmap");
        exit(1);
    }

    res.buf = hugepage_addr;
    memset(hugepage_addr, 0, BUF_SIZE);

    printf("hugepage starting address is: %p\n", hugepage_addr);
    printf("socket connection\n");
    rc = make_two_tcp_connection(&res, &config);
    if (rc < 0) {
        perror("sock connect");
        exit(1);
    }


    verify_fd = open(argv[2], O_RDONLY);
    if (verify_fd < 0) {
        perror("open");
        exit(1);
    }

    verify_matrix = (double *) mmap(0, M * N * K * sizeof(double), PROT_READ, MAP_PRIVATE, verify_fd, 0);
    if (verify_matrix==MAP_FAILED) {
        perror("mmap");
        exit(1);
    }

    printf("calculating the result of pagerank\n");
    rc = nds_tensor_verify(&res, matrix_id, M, SUB_M, verify_matrix);
    
    close(res.sock);
    close(res.req_sock);
    munmap(hugepage_addr, BUF_SIZE);
    close(hugepage_fd);

    munmap(verify_matrix, M * N * K * sizeof(double));
    close(verify_fd);
    return rc;
}
